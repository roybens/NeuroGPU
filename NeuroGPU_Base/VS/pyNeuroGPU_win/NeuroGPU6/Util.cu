#include "hip/hip_runtime.h"
#include "Util.h"
//#include "CudaStuff.cuh"
#define checkCudaErrors(val)           check ( (val), #val, __FILE__, __LINE__ )
MYFTYPE  maxf(MYFTYPE  a, MYFTYPE  b) {
	if (a>b)
		return a;
	return b;
}

MYFTYPE  MaxAbsDiffVec(MYFTYPE * A, MYFTYPE  *B, int n) {
	MYFTYPE  cmax = -1;
	MYFTYPE  curdiff;
	for (int i = 0; i<n; i++) {
		curdiff = abs(A[i] - B[i]);
		cmax = maxf(cmax, curdiff);
	}
	return cmax;
}

void CopyVec(MYFTYPE * A, MYFTYPE  *B, int n) {
	for (int i = 0; i<n; i++) {
		A[i] = B[i];
	}
}
void CopyVecMYSECONDFTYPE(MYSECONDFTYPE* A, MYSECONDFTYPE  *B, int n) {
	for (int i = 0; i<n; i++) {
		A[i] = B[i];
	}
}
void CopyVecTwoTypes(MYSECONDFTYPE *A, MYFTYPE * B, int n) {
	for (int i = 0; i<n; i++) {
		A[i] = B[i];
	}
}
void FlipVec(MYFTYPE * A, MYFTYPE  *B, int n) {
	for (int i = 0; i<n; i++) {
		A[i] = B[n - i];
	}
}
void ReadShortFromCSV(char* line, short *ans, int n) {
	int count = 0;
	char* tok;
	unsigned short tmp;
	if (n == 1) {
		tok = strtok(line, ",");
		//printf("%s\n", tok);
		tmp = atoi(tok);
		*ans = (short)tmp;
	}
	else {
	
		for (tok = strtok(line, ","); tok != NULL; tok = strtok(NULL, ",")){
		if(count<n){
		
			tmp = atoi(tok);
			ans[count++] = (short)tmp;
			//printf("%d-%d,-%s,** %c,%c\n",count,strlen(tok),tok,tok[0],tok[1]);

		}
		}
		//printf("read short %d\n",count);
		}
	

}
void ReadIntFromCSV(char* line, int *ans, int n) {
	int count = 0;
	char* tok;
	int tmp;
	if (n == 1) {
		tok = strtok(line, ",");
		//printf("%s\n", tok);
		tmp = atoi(tok);
		*ans = (int)tmp;
	}
	else {
	     
		for (tok = strtok(line, ","); tok != NULL; tok = strtok(NULL, ","))
		{
			tmp = atoi(tok);
			ans[count++] = (int)tmp;

		}
		}
	
}


void ReadDoubleFromCSV(char* line, double *ans, int n) {
	int count = 0;
	char* tok;
	double tmp;
	if (n == 1) {
		tok = strtok(line, ",");
		tmp = atof(tok);
		*ans = (double)tmp;
	}
	else {

		for (tok = strtok(line, ","); tok != NULL; tok = strtok(NULL, ","))
		{
			if (tok[1] != '\n'){
						tmp = atof(tok);
					//	printf("%d-%d,-%s,** %c,%c\n",count,strlen(tok),tok,tok[0],tok[1]);
						ans[count++] = tmp;
						}

		}
	}
	//printf("\ndone with readdouble\n");
}
void ReadFloatFromCSV(char* line, MYFTYPE *ans, int n) {
	int count = 0;
	char* tok;
	MYFTYPE tmp;
	if (n == 1) {
		tok = strtok(line, ",");
		tmp = atof(tok);
		*ans = (MYFTYPE)tmp;
	}
	else {

		for (tok = strtok(line, ","); tok != NULL; tok = strtok(NULL, ","))
		{
		if (n>count){
			tmp = atof(tok);
			//printf("%d-%d,-%s,** %c,%c",count,strlen(tok),tok,tok[0],tok[1]);
			ans[count++] = tmp;
			}

		}
	}
}
int power(int base, int exp)
{
	int power;
	power = 1;
	while (exp-- > 0)
		power *= base;

	return power;
}
MYFTYPE myatof(char s[])
{
	MYFTYPE val, pow;
	int sign, i, esign, exp;
	int power(int base, int exp);

	for (i = 0; isspace(s[i]); i++)
		;

	sign = (s[i] == '-') ? -1 : 1;

	if (s[i] == '+' || s[i] == '-')
		i++;

	for (val = 0.0; isdigit(s[i]); i++)
		val = 10.0 * val + (s[i] - '0');

	if (s[i] == '.')
		i++;

	for (pow = 1.0; isdigit(s[i]); i++)
	{
		val = 10.0 * val + (s[i] - '0');
		pow *= 10.0;
	}

	if (s[i] == 'e' || s[i] == 'E')
		i++;
	if (s[i] == '+' || s[i] == '-')
	{
		esign = s[i];
		i++;
	}

	for (exp = 0; isdigit(s[i]); i++)
		exp = 10.0 * exp + (s[i] - '0');

	if (esign == '-')
		return sign * (val / pow) / power(10, exp);
	else

		return sign * (val / pow) * power(10, exp);
}

void ReadFloatWithEFromCSV(char* line, MYFTYPE *ans, int n) {
	int count = 0;
	char* tok;
	MYFTYPE tmp;
	if (n == 1) {
		tok = strtok(line, ",");
		tmp = atof(tok);
		*ans = (MYFTYPE)tmp;
	}
	else {

		for (tok = strtok(line, ","); (tok != NULL && tok[0] != '\n'); tok = strtok(NULL, ","))
		{
			if (n>count){
			tmp = myatof(tok);
			ans[count++] = tmp;
			}

		}
	}
}


MYFTYPE* ReadAllParams(const char* FN, MYDTYPE NParams, MYDTYPE Nx, int  &nSets) {
	MYFTYPE* ans;
	//printf("readingggg params %s\n",FN);
	FILE *fl = fopen(FN, "r"); // YYY add FILE*
	if (!fl) {
		printf("Failed to read allparmas.csv\n");
		return NULL;
	}
	char line[1009600];
	fgets(line, sizeof(line), fl);
	ReadIntFromCSV(line, &nSets, 1);
	//printf("reading params nsets is %d\n",nSets);
	ans = (MYFTYPE *)malloc(Nx * NParams * nSets * sizeof(MYFTYPE));
	//printf("Nx %d nparams %d,nSets%d\n",Nx,NParams,nSets);
	//printf("malloc size is %d\n",Nx*NParams*nSets*sizeof(MYFTYPE));
	for (int i = 0; i<nSets; i++) {
		fgets(line, sizeof(line), fl);
		//printf("a%d.adreess is %d",i,i*Nx*NParams);
		ReadFloatWithEFromCSV(line, &ans[i*Nx*NParams], Nx*NParams);
		//printf("\n");
	}
	//printf("done filling params");
	fclose(fl);
	return ans;
}
MYFTYPE* ReadInitStates(const char *FN, MYDTYPE NStates, MYDTYPE Nx, MYDTYPE  nSets) {
	/* TODO: edit this file opening stuff based on how the init states file is named */
	MYFTYPE* ans;
	MYDTYPE nsetsFromFile;
	FILE *fl = fopen(FN, "r"); // YYY add FILE*
	if (!fl) {
		printf("Failed to read allStates.csv\n");
		return NULL;
	}
	char line[1009600];
	fgets(line, sizeof(line), fl);
	ReadShortFromCSV(line, &nsetsFromFile, 1);
	//printf("nsetsfrom file is %d", nsetsFromFile);
	if (nsetsFromFile != nSets)
		printf("we have a problem nsets from params does not much states");
	ans = (MYFTYPE *)malloc(Nx * NStates * nSets * sizeof(MYFTYPE));
	for (int i = 0; i<nSets; i++) {
		fgets(line, sizeof(line), fl);
		ReadFloatWithEFromCSV(line, &ans[i*Nx*NStates], Nx*NStates);
	}
	fclose(fl);
	return ans;
}



void ReadStimData(const char* FN, Stim &stim, MYDTYPE Nx) {
	char FileName[300];
	//sprintf(FileName,"%s%d.dat",FN,MUL32*32);
    
	printf("Start reading file - %s ReadStimData() \n",FileName);

	FILE *fl;
	fl = fopen(FileName, "rb");
	if (!fl)
	{
		printf("Failed to read StimData\n");
		return;
	}
	//fread(&stim.NStimuli, sizeof(MYDTYPE),1, fl);
	stim.NStimuli = 1;
	stim.dels = (MYFTYPE*)malloc(stim.NStimuli * sizeof(MYFTYPE));
	fread(stim.dels, sizeof(MYFTYPE), stim.NStimuli, fl);
	stim.durs = (MYFTYPE*)malloc(stim.NStimuli * sizeof(MYFTYPE));
	fread(stim.durs, sizeof(MYFTYPE), stim.NStimuli, fl);
	stim.amps = (MYFTYPE*)malloc(stim.NStimuli * sizeof(MYFTYPE));
	fread(stim.amps, sizeof(MYFTYPE), stim.NStimuli, fl);
	fread(&stim.comp, sizeof(MYDTYPE), 1, fl);
	fread(&stim.area, sizeof(MYFTYPE), 1, fl);
	fread(&stim.loc, sizeof(MYDTYPE), 1, fl);
	//stim.loc = stim.loc - 1;
	fclose(fl);
	return;
}




void CreateStimData(Stim &stim) {
	stim.NStimuli = NSTIM;
	stim.dels = (MYFTYPE*)malloc(stim.NStimuli * sizeof(MYFTYPE));
	stim.durs = (MYFTYPE*)malloc(stim.NStimuli * sizeof(MYFTYPE));
	stim.amps = (MYFTYPE*)malloc(stim.NStimuli * sizeof(MYFTYPE));
	stim.comp = 1;
	stim.area = 28.3771;
	stim.loc = 2;
	MYFTYPE  initAmp = -1;
	for (int i = 0; i<stim.NStimuli; i++) {
		stim.durs[i] = 20.0;
		stim.dels[i] = 10.0;
		stim.amps[i] = initAmp + 1 * i;
	}
	return;
}

void ReadStimFromFile(const char* FN, Stim &stim) {//deprecated
	FILE *fl;
	fl = fopen(FN, "rb");
	printf("reading stimfromfile - %s", FN);
	if (!fl)
	{
		printf("Failed to read StimData\n");
		return;
	}
	MYDTYPE Nt;
	fread(&stim.NStimuli, sizeof(MYDTYPE), 1, fl);
	fread(&Nt, sizeof(MYDTYPE), 1, fl);
	stim.Nt = MYDTYPE(ceil(MYFTYPE(Nt) / WARPSIZE)*WARPSIZE);
	fread(&stim.comp, sizeof(MYDTYPE), 1, fl);
	fread(&stim.loc, sizeof(MYDTYPE), 1, fl);
	stim.loc -= 1;
	fread(&stim.area, sizeof(MYFTYPE), 1, fl);
	fread(&stim.numofdts, sizeof(MYDTYPE), 1, fl);
	stim.dtInds = (MYDTYPE*)malloc(stim.numofdts * sizeof(MYDTYPE));//dont need it now
	stim.durs = (MYFTYPE*)malloc(stim.numofdts * sizeof(MYFTYPE));//acts as dts.
	stim.amps = (MYFTYPE*)malloc(Nt*stim.NStimuli * sizeof(MYFTYPE));
	fread(stim.dtInds, sizeof(MYDTYPE), stim.numofdts, fl);
	fread(stim.durs, sizeof(MYFTYPE), stim.numofdts, fl);
	fread(stim.amps, sizeof(MYFTYPE), Nt*stim.NStimuli, fl);
	fclose(fl);

}
void ReadCSVStim(Stim &stim,int argc) {
	char FileName[300];
	sprintf(FileName, "%s", Stim_csv_meta);
	//sprintf(FileName,"%s%d.dat",FN,MUL32*32);
	FILE *fl = fopen(FileName, "r"); // YYY add FILE*
	if (!fl) {
		printf("Failed to read metaStimData1\n");

	}
	char line[160000];
	fgets(line, sizeof(line), fl);
	ReadShortFromCSV(line, &stim.NStimuli, 1);
	fgets(line, sizeof(line), fl);
	ReadFloatFromCSV(line, &stim.Nt, 1);
	fgets(line, sizeof(line), fl);
	ReadShortFromCSV(line, &stim.comp, 1);//this is not currently used.
	fgets(line, sizeof(line), fl);
	ReadShortFromCSV(line, &stim.loc, 1);
	stim.loc -= 1;
	fgets(line, sizeof(line), fl);
	ReadFloatFromCSV(line, &stim.area, 1);
    int stim_ind;
    hipGetDevice(&stim_ind);
	if (argc > 1) {
		sprintf(FileName, "%s%d.csv", Stim_csv_raw, stim_ind);
	}
	else {
		sprintf(FileName, "%s.csv", Stim_csv_raw);
	}
	FILE *f2 = fopen(FileName, "r");
	if (!f2) {
		printf("Failed to read StimRaw Data2 - %s\n",FileName);

	}

	stim.amps = (MYFTYPE*)malloc(stim.Nt*stim.NStimuli * sizeof(MYFTYPE));
	for (int i = 0; i < stim.NStimuli; i++) {

		fgets(line, sizeof(line), f2);
		ReadFloatFromCSV(line, &stim.amps[i*int(stim.Nt)], stim.Nt);
	}

	sprintf(FileName, "%s", Time_steps_FN);
	//sprintf(FileName,"%s%d.dat",FN,MUL32*32);
	FILE *f3 = fopen(FileName, "r"); // YYY add FILE*
	if (!f3) {
		printf("Failed to read SimData3\n");

	}
	stim.durs = (MYFTYPE*)malloc(stim.Nt * sizeof(MYFTYPE));
	fgets(line, sizeof(line), f3);

	ReadFloatFromCSV(line, stim.durs, stim.Nt);
}
void FreeStimData(Stim &stim) {
	printf("Still need to implement FreeStimData()\n");
}
void ReadSimData(const char* FN, MYDTYPE N, Sim &sim) {
	char FileName[300];
	sprintf(FileName, "%s.csv", FN);
	//sprintf(FileName,"%s%d.dat",FN,MUL32*32);
	FILE *fl = fopen(FileName, "r"); // YYY add FILE*
	if (!fl) {
		printf("Failed to read SimData4\n");

	}

	char line[1009600];


	printf("Start reading file - ReadSimData()\n");
	MYFTYPE v_init;
	fgets(line, sizeof(line), fl);
	ReadFloatFromCSV(line, &v_init, 1);

	sim.Vs = (MYFTYPE*)malloc(N * sizeof(MYFTYPE));
	for (int i = 0; i < N; i += 1) {
		sim.Vs[i] = v_init;
	}
	fgets(line, sizeof(line), fl);
	ReadFloatFromCSV(line, &sim.dt, 1);
	//fgets(line, sizeof(line), fl);
	//ReadFloatFromCSV(line, &sim.TFinal, 1);
	//MYDTYPE Nt = ceil(sim.TFinal / sim.dt);
	//MYDTYPE to32 = 32 - Nt % 32;
	//sim.TFinal = sim.TFinal + (to32 + 1)*sim.dt;
	fgets(line, sizeof(line), fl);
	ReadShortFromCSV(line, &sim.NRecSites, 1);
	sim.RecSites = (MYDTYPE*)malloc(sim.NRecSites * sizeof(MYDTYPE));
	fread(sim.RecSites, sizeof(MYDTYPE), sim.NRecSites, fl);
	//fread(&sim.Celsius, sizeof(MYFTYPE),1, fl);
	fgets(line, sizeof(line), fl);
	ReadShortFromCSV(line, sim.RecSites, sim.NRecSites);
	fclose(fl);
	return;
}
void FreeSimData(Sim &sim) {
	printf("Still need to implement FreeStimData()\n");
}
double diffclock(clock_t clock1, clock_t clock2)
{
	double diffticks = clock1 - clock2;
	double diffms = (diffticks) / CLOCKS_PER_SEC;
	return diffms;
}

void SaveArrayToFile(const char* FN, const int N, const double* Arr) {
	printf("printing %s size is %d\n", FN, N);
	const int prec = 3;
	
	FILE *file = fopen(FN, "wb");
	if (file) {
		fwrite(&N, sizeof(int), 1, file);
		fwrite(&prec, sizeof(int), 1, file);
		fwrite(Arr, sizeof(double), N, file);
	}
	else {
		printf("ERR SaveArrayToFile %s %d\n", FN, N);
	}
	fclose(file);
}
void SaveArrayToFile(const char* FN, const int N, const float* Arr) {
	printf("\nprinting %s size is %d\n", FN, N);
	double* arr_dbl;
	arr_dbl =(double*) malloc(N * sizeof(double));
	const int prec = 3;
	for (int i = 0; i < N; i++) {
		arr_dbl[i] = (double)Arr[i];
	}
	FILE *file = fopen(FN, "wb");
	if (file) {
		fwrite(&N, sizeof(int), 1, file);
		fwrite(&prec, sizeof(int), 1, file);
		fwrite(arr_dbl, sizeof(double), N, file);
	}
	else {
		printf("ERR SaveArrayToFile %s %d\n", FN, N);
	}
	fclose(file);
}

MYFTYPE* transposeMat(MYFTYPE* Arr, MYDTYPE width, MYDTYPE length) {
	MYFTYPE* ans = (MYFTYPE*)malloc(width*length * sizeof(MYFTYPE));
	for (int row = 0; row < length; row++) {
		for (int col = 0; col < width; col++) {
			ans[col*length + row] = Arr[row*width + col];
		}
	}
	return ans;
}

void CompareArrays(const MYFTYPE * App, const MYFTYPE  *Real, int n, char* Prefix) {
	MYFTYPE MaxDiff = 0, CurDiff, MaxRDiff = 0;
	for (int i = 0; i<n; i++) {
		CurDiff = abs(App[i] - Real[i]);
		MaxDiff = maxf(MaxDiff, CurDiff);
		MaxRDiff = maxf(MaxRDiff, CurDiff / abs(Real[i]));
	}
	printf("Comparing %s,%d: MaxD=%f, MaxRD=%f\n", Prefix, n, MaxDiff, MaxRDiff);
}
void SaveVHots(const char* FN, MYFTYPE** VHotsHost, MYDTYPE Nt, MYDTYPE NStimuli) {
	FILE *file = fopen(FN, "wb");
	if (file) {
		for (int i = 0; i<NStimuli; i++) {
			fwrite(VHotsHost[i], sizeof(MYFTYPE), Nt, file);
		}
	}
	else {
		printf("ERR SaveArrayToFile %s %d\n", FN);
	}
	fclose(file);
}
/*
void ReadDebugData(const char* FN, MYFTYPE** DebugData, MYDTYPE Nsegs, MYDTYPE Nt){
FILE *fl;
MYDTYPE NP;
fl = fopen(FN, "rb");

for (int i = 0; i<NSTATES; i++){
DebugData[i] = (MYFTYPE*)malloc((Nsegs*Nt)*sizeof(MYFTYPE));
fread(DebugData[i], sizeof(MYFTYPE), Nsegs*Nt, fl);
}
}

void SetStatesFromDebug(MYFTYPE** StatesM, MYFTYPE** DebugData, MYDTYPE iter, MYDTYPE Nseg){
MYDTYPE debugIter;
if (iter>0){
iter -= 1;
debugIter = iter*Nseg;
for (int currSeg = 0; currSeg<Nseg; currSeg++){
for (int currState = 0; currState<NSTATES; currState++){
StatesM[currState][currSeg] = DebugData[currState][debugIter + currSeg];
}
}
}
}
*/
void ReadRHSData(const char* FN, MYFTYPE** DebugData, MYDTYPE Nsegs, MYDTYPE Nt) {
	FILE *fl;
	fl = fopen(FN, "rb");
	for (int i = 0; i<Nt; i++) {
		DebugData[i] = (MYFTYPE*)malloc((Nsegs) * sizeof(MYFTYPE));
		fread(DebugData[i], sizeof(MYFTYPE), Nsegs, fl);
	}
}

void SetRHSFromNeuron(MYFTYPE* rhs, MYFTYPE** DebugData, MYDTYPE i, MYDTYPE N) {

	for (int currSeg = 0; currSeg<N; currSeg++) {
		rhs[currSeg] = DebugData[i][currSeg];
	}


}
void ReadDData(const char* FN, MYFTYPE** DebugData, MYDTYPE Nsegs, MYDTYPE Nt) {
	FILE *fl;
	fl = fopen(FN, "rb");
	for (int i = 0; i<Nt; i++) {
		DebugData[i] = (MYFTYPE*)malloc((Nsegs) * sizeof(MYFTYPE));
		fread(DebugData[i], sizeof(MYFTYPE), Nsegs, fl);
	}
}

void SetDFromNeuron(MYFTYPE* D, MYFTYPE** DebugData, MYDTYPE i, MYDTYPE N) {

	for (int currSeg = 0; currSeg<N; currSeg++) {
		D[currSeg] = DebugData[i][currSeg];

	}

}
void ReadVData(const char* FN, MYFTYPE** DebugData, MYDTYPE Nsegs, MYDTYPE Nt) {
	FILE *fl;
	fl = fopen(FN, "rb");
	for (int i = 0; i<Nt; i++) {
		DebugData[i] = (MYFTYPE*)malloc((Nsegs) * sizeof(MYFTYPE));
		fread(DebugData[i], sizeof(MYFTYPE), Nsegs, fl);
	}
}
void SetVFromNeuron(MYFTYPE* vs, MYFTYPE** DebugData, MYDTYPE i, MYDTYPE N) {
	if (i>0) {
		for (int currSeg = 0; currSeg<N; currSeg++) {
			vs[currSeg] = DebugData[i - 1][currSeg];
		}
	}

}
void debugPrintMYFTYPE(MYFTYPE* A, MYDTYPE N, FILE* fdebug) {
	fwrite(&N, sizeof(MYDTYPE), 1, fdebug);
	fwrite(A, sizeof(MYFTYPE), N, fdebug);
}
void debugPrintMYSECONDFTYPE(MYSECONDFTYPE* A, MYDTYPE N, FILE* fdebug) {
	fwrite(&N, sizeof(MYDTYPE), 1, fdebug);
	fwrite(A, sizeof(MYSECONDFTYPE), N, fdebug);
}
bool IsGPUCapableP2P(hipDeviceProp_t *pProp) {
#ifdef _WIN32
	return (bool)(pProp->tccDriver ? true : false);
#else
	return (bool)(pProp->major >= 2);
#endif
}


bool IsAppBuiltAs64()
{
	return sizeof(void*) == 8;
}
int* checkPeerAccess(int &n_p2p) {
	n_p2p = 0;
	//printf("[%s] - Starting...\n");
	if (!IsAppBuiltAs64())
	{
	//	printf("%s is only supported with on 64-bit OSs and the application must be built as a 64-bit target.  Test is being waived.\n");
	}
	// Number of GPUs
	//printf("Checking for multiple GPUs...\n");
	int gpu_n;
	CUDA_RT_CALL(hipGetDeviceCount(&gpu_n));
	printf("CUDA-capable device count: %i\n", gpu_n);
	if (gpu_n < 2)
	{
		printf("Two or more GPUs with SM 2.0 or higher capability are required for %s.\n");
		printf("Waiving test.\n");

	}
	// Query device properties
	hipDeviceProp_t prop[64];
	int gpuid[64]; // we want to find the first two GPU's that can support P2P
	int gpu_count = 0;   // GPUs that meet the criteria
	for (int i = 0; i < gpu_n; i++)
	{
		CUDA_RT_CALL(hipGetDeviceProperties(&prop[i], i));

		// Only boards based on Fermi can support P2P
		if ((prop[i].major >= 2)
#ifdef _WIN32
			// on Windows (64-bit), the Tesla Compute Cluster driver for windows must be enabled
			&& prop[i].tccDriver
#endif
			)
		{
			// This is an array of P2P capable GPUs
			gpuid[gpu_count++] = i;
		}

	//	printf("> GPU%d = \"%15s\" %s capable of Peer-to-Peer (P2P)\n", i, prop[i].name, (IsGPUCapableP2P(&prop[i]) ? "IS " : "NOT"));
	}

	// Check for TCC for Windows
	if (gpu_count < 2)
	{
		printf("\nTwo or more GPUs with SM 2.0 or higher capability are required for %s.\n");
#ifdef _WIN32
		printf("\nAlso, a TCC driver must be installed and enabled to run %s.\n");
#endif
		CUDA_RT_CALL(hipSetDevice(0));


	}
	// Check possibility for peer access
	//printf("\nChecking GPU(s) for support of peer to peer memory access...\n");
	int can_access_peer;
	int* p2pCapableGPUs; // We take only 1 pair of P2P capable GPUs
	p2pCapableGPUs = (int*)malloc(gpu_n * sizeof(int));


	p2pCapableGPUs[0] = 0; //checking who has access to 0 and 0 has peeracces to himself
						   // Show all the combinations of supported P2P GPUs
	for (int i = 1; i < gpu_count; i++)
	{
		p2pCapableGPUs[i] = -1;
		CUDA_RT_CALL(hipDeviceCanAccessPeer(&can_access_peer, gpuid[i], 0));
	//	printf("> Peer access from %s (GPU%d) -> %s (GPU%d) : %s\n", prop[gpuid[i]].name, gpuid[i],prop[0].name, gpuid[0],can_access_peer ? "Yes" : "No");
		if (can_access_peer)
		{
			p2pCapableGPUs[i] = gpuid[i];
			n_p2p += 1;

		}
	}



	return p2pCapableGPUs;
}

void enablePeerAccess(int* p2pCapableGPUs, int np2p) {
	hipDeviceProp_t prop[64];
	int gpuid[64];
	// Use all  of p2p to 0 capable GPUs detected.
	int gpu_n;
	//int index = 0;
	CUDA_RT_CALL(hipGetDeviceCount(&gpu_n));
	for (int i = 1; i < np2p; i++) {
	//	printf("in enablep2p i is %d p2p is %d\n", i, np2p);
		gpuid[i] = p2pCapableGPUs[i];
		CUDA_RT_CALL(hipGetDeviceProperties(&prop[i], gpuid[i]));
	//	printf("Enabling peer access from GPU%d to GPU%d...\n", gpuid[i], 0);
		CUDA_RT_CALL(hipSetDevice(gpuid[i]));
		CUDA_RT_CALL(hipDeviceEnablePeerAccess(0, 0));
	//	printf("Checking GPU%d and GPU%d for UVA capabilities...\n", gpuid[0], gpuid[i]);
		const bool has_uva = (prop[gpuid[0]].unifiedAddressing && prop[gpuid[i]].unifiedAddressing);
	//	printf("> %s (GPU%d) supports UVA: %s\n", prop[gpuid[i]].name, gpuid[i], (prop[gpuid[i]].unifiedAddressing ? "Yes" : "No"));
	}

	//printf("leaving enablepeeraccess");
}
int char2int(char* str){
return atoi(str);
}