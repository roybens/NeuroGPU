#include "hip/hip_runtime.h"
#include "CudaStuff.cuh"
//#include "AllModels.cu"
#include "AllModels.cuh"
#define ILP16
__constant__ MYFTYPE cCm[NSEG];
__constant__ MYSECONDFTYPE cE[NSEG];
__constant__ MYSECONDFTYPE cF[NSEG];
__constant__ MYDTYPE cFIdxs[NSEG*LOG_N_DEPTH];
__constant__ MYDTYPE cKs[NSEG];
__constant__ MYDTYPE cSegToComp[NSEG];
__constant__ MYDTYPE cBoolModel[NSEG * N_MODELS];//One day change this to bool
__constant__ MYDTYPE cRelStarts[N_FATHERS];//nFathers
__constant__ MYDTYPE cRelEnds[N_FATHERS];//nFathers
__constant__ MYDTYPE cFathers[N_FATHERS];//nFathers
__constant__ MYDTYPE cRelVec[N_CALL_FOR_FATHER];//nCallForFather
__constant__ MYDTYPE cSegStartI[N_CALL_FOR_FATHER + 1];//nCallForFather
__constant__ MYDTYPE cSegEndI[N_CALL_FOR_FATHER + 1];//nCallForFather

__constant__ MYDTYPE cCompByLevel32[COMP_DEPTH*WARPSIZE];//CompDepth
__constant__ MYDTYPE cCompByFLevel32[COMP_DEPTH*WARPSIZE];//CompFDepth
__constant__ MYDTYPE cLRelStarts[N_L_REL];//nLRel
__constant__ MYDTYPE cLRelEnds[N_L_REL];//nLRel
__constant__ MYDTYPE cFLRelStarts[N_F_L_REL];//nFLRel
__constant__ MYDTYPE cFLRelEnds[N_F_L_REL];//nFLRel
__constant__ MYDTYPE cSonNoVec[NSEG];//InMat.N 


#ifdef __INTELLISENSE__
void __syncthreads();
#endif
__device__ void BeforeLU(HMat InMat, MYSECONDFTYPE* uHP, MYSECONDFTYPE* bHP, MYDTYPE Depth)
{
	MYDTYPE PIdx = threadIdx.x;
	MYDTYPE i, j, CurJ, CurB, t, CurLevel, LRelIndex;
	MYDTYPE JumctionI;
	LRelIndex = cLRelStarts[CurLevel];
	LRelIndex = LRelIndex + cLRelEnds[CurLevel];
	for (CurLevel = 0; CurLevel <= Depth; CurLevel++) {

		for (LRelIndex = cLRelStarts[CurLevel]; LRelIndex <= cLRelEnds[CurLevel]; LRelIndex++) {
			//for(LRelIndex=cLRelStarts[CurLevel];LRelIndex<=InMat.LRelEnds[CurLevel];LRelIndex++){ 
			JumctionI = cCompByLevel32[LRelIndex*WARPSIZE + PIdx] - 1;
			for (i = cSegStartI[JumctionI] - 1; i<cSegEndI[JumctionI]; i++) {
				MYSECONDFTYPE uHPm1 = uHP[i - 1];

				uHP[i] = uHP[i] - cF[i - 1] * (cE[i - 1] / uHPm1); // So far same as paper parallel
				uHPm1 = uHP[i - 1];
				MYSECONDFTYPE bHPm1 = bHP[i - 1];
				bHP[i] = bHP[i] - bHPm1*cE[i - 1] / uHPm1; // bH is y
			}
		}
		if (CurLevel<Depth) {
			for (LRelIndex = cFLRelStarts[CurLevel]; LRelIndex <= cFLRelEnds[CurLevel]; LRelIndex++) {
				CurB = cCompByFLevel32[(LRelIndex)*WARPSIZE + PIdx] - 1;//RB i inserted another  -1 into the index RB 2 i removed the-1 from the curlevel 
				CurJ = cFathers[CurB] - 1;
				MYDTYPE St = cRelStarts[CurB];
				MYDTYPE En = cRelEnds[CurB];
				for (j = St; j <= En; j++) {
					t = cRelVec[j - 1] - 1;
					MYSECONDFTYPE uHPm1 = uHP[t - 1];
					uHP[CurJ] -= cF[t - 1] * (cE[t - 1] / uHPm1);
					uHPm1 = uHP[t - 1];
					MYSECONDFTYPE bHPm1 = bHP[t - 1];
					bHP[CurJ] -= bHPm1*cE[t - 1] / uHPm1;
				}
			}
		}
	}
}

#ifdef BKSUB1
__device__ void BkSub(HMat InMat, MYSECONDFTYPE* PX, MYSECONDFTYPE* PF, MYSECONDFTYPE* uHP, MYSECONDFTYPE* bHP, MYDTYPE LognDepth)
{
	// MYDTYPE PIdx_1=threadIdx.x; // MYDTYPE NextID_1;
	MYDTYPE PIdx[NILP + 1];
	MYDTYPE NextID[NILP + 1];
	for (int count = 1; count < NILP + 1; count++) {
		PIdx[count] = threadIdx.x + (WARPSIZE*(count - 1));
		//this is from a different superilp
		PX[PIdx[count]] = PX[PIdx[count]] / PF[PIdx[count]];
		PF[PIdx[count]] = -cF[PIdx[count]] / PF[PIdx[count]];
	}
	MYFTYPE OldPXj[NILP + 1];
	MYFTYPE OldPXNextID[NILP + 1];
	MYFTYPE OldPFj[NILP + 1];
	MYFTYPE OldPFNextID[NILP + 1];
	MYDTYPE i;
	PX[InMat.N] = 0;
	PF[InMat.N] = 1;
	for (i = 0; i<LognDepth; i++) {
		for (int count = 1; count < NILP + 1; count++) {
			NextID[count] = cFIdxs[i*InMat.N + PIdx[count]] - 1;
			OldPXj[count] = PX[PIdx[count]];
			OldPXNextID[count] = PX[NextID[count]];
			PX[PIdx[count]] = OldPXj[count] + OldPXNextID[count] * PF[PIdx[count]];
		}
		for (int count = 1; count < NILP + 1; count++) {
			OldPFj[count] = PF[PIdx[count]];
			OldPFNextID[count] = PF[NextID[count]];
			PF[PIdx[count]] = OldPFj[count] * OldPFNextID[count];
		}
	}
}
#endif

#ifdef BKSUB2
__device__ void BkSub(HMat InMat, MYSECONDFTYPE* uHP, MYSECONDFTYPE* bHP, MYSECONDFTYPE* Out, MYDTYPE Depth)
{
	// MYDTYPE PIdx_3=threadIdx.x+(WARPSIZE*2); Out[PIdx_2]=0; // might be useless?
#define THISCOMMANDHEREB1(VARILP) MYDTYPE PIdx_ ## VARILP =threadIdx.x+(WARPSIZE*( ## VARILP -1)); Out[PIdx_ ## VARILP ]=0;
	MYDTYPE PIdx[1] = threadIdx.x + (WARPSIZE*([1] - 1)); Out[PIdx[1]] = 0;

	MYDTYPE j, CurJ, CurB, t;
	MYDTYPE JumctionI;
	short CurLevel, i;

	// get KsB from MATLAB (this comes instead of FIdxsX)
	// KsB=Ks; // do in matlab

	// bPX=zeros(1,N); // might be useless
	// for CurLevel=Depth:-1:0
	MYDTYPE LRelIndex, k;
	MYFTYPE temp;
	for (CurLevel = Depth; CurLevel >= 0; CurLevel--) {
		//     Run all independent set for this level, in parallel
		// for JumctionI=find(Level==CurLevel) % in parallel
		for (LRelIndex = cLRelStarts[CurLevel]; LRelIndex <= cLRelEnds[CurLevel]; LRelIndex++) {
			JumctionI = cCompByLevel32[LRelIndex*WARPSIZE + PIdx_1] - 1;
			// for i=(cSegEndI(JumctionI)):-1:(cSegStartI(JumctionI)-1)
			for (i = cSegEndI[JumctionI] - 1; i >= (cSegStartI[JumctionI] - 2); i--) {
				// k=cKsB(i+1);
				k = cKsB[i + 1];

				// bPX(i)=(bH(i)-bPX(k)*f(i))/uH(i);
				Out[i] = (bHP[i] - Out[k] * cF[i]) / uHP[i];
			}
		}
	}
}
#endif

__device__ void runSimulation(HMat InMat, const MYFTYPE* __restrict__ ParamsM, MYFTYPE* ModelStates, MYFTYPE* V, Stim stim, Sim sim, MYFTYPE* VHotGlobal)
{

	__shared__ MYSECONDFTYPE uHP_all[(NSEG + 2)*NTRACES];
	__shared__ MYSECONDFTYPE bHP_all[(NSEG + 2)*NTRACES];
	__shared__ MYFTYPE SMemVHot_all[WARPSIZE*NTRACES];
	MYSECONDFTYPE *uHP = &uHP_all[(NSEG + 2)*threadIdx.y];
	MYSECONDFTYPE *bHP = &bHP_all[(NSEG + 2)*threadIdx.y];
	MYFTYPE *SMemVHot = &SMemVHot_all[(WARPSIZE)*threadIdx.y];
	MYDTYPE StimID = threadIdx.y;

	//MYDTYPE PerStimulus;
	//PerStimulus = InMat.N+2;
	
	
	
	MYDTYPE NeuronID = blockIdx.x;
	int Nt = stim.Nt;
	MYFTYPE t = 0;
	MYSECONDFTYPE *PX, *PF;
	PX = bHP;
	PF = uHP;
	MYDTYPE PIdx[NILP + 1];
	for (int count = 1; count < NILP + 1; count++) {
		PIdx[count] = threadIdx.x + (WARPSIZE*(count - 1));
	}
	int perBlockStatesSize = (NSEG)*(NSTATES + 1);
#define state_macro(stateind,segmentInd) ModelStates[NeuronID*perBlockStatesSize + stateind*NSEG+PIdx[segmentInd]]//Is this coalesced?
	MYFTYPE Vmid[NILP + 1];
	MYFTYPE v[NILP + 1];
	MYFTYPE dv[NILP + 1];
	MYSECONDFTYPE sumCurrents[NILP + 1];
	MYSECONDFTYPE sumCurrentsDv[NILP + 1];
	MYFTYPE sumConductivity[NILP + 1];
	MYFTYPE sumConductivityDv[NILP + 1];
	MYDTYPE parentIndex[NILP + 1];
	MYDTYPE Eidx[NILP + 1];
	MYSECONDFTYPE rhs[NILP + 1];
	MYSECONDFTYPE D[NILP + 1];
	MYFTYPE gModel[NILP + 1];
	MYFTYPE cai[NILP + 1];
	MYFTYPE ica[NILP + 1];
	MYFTYPE eca[NILP + 1];
	MYFTYPE StimCurrent[NILP + 1];
	for (int count = 1; count < NILP + 1; count++) {
		v[count] = V[PIdx[count]];
		sumCurrents[count] = 0;
		sumCurrentsDv[count] = 0;
		sumConductivity[count] = 0;
		sumConductivityDv[count] = 0;
		bHP[count] = 0;
		dv[count] = 0;
		Eidx[count] = InMat.N - PIdx[count] - 1;
		parentIndex[count] = InMat.N - cKs[InMat.N - PIdx[count]];
		cai[count] = 0;
		ica[count] = 0;
		eca[count] = 0;
		if (PIdx[count] == 0) {
			parentIndex[count] = 0;
		};
		for (int count1 = 0; count1 < NSTATES; count1++) {
			state_macro(count1, count) = 0;
		}
	}

	if (Eidx[1]>InMat.N - 1) {
		Eidx[1] = InMat.N - 1;
	}
	MYDTYPE perThreadParamMSize = InMat.NComps*NPARAMS;

#define param_macro(paramInd,segmentInd) ParamsM[NeuronID*perThreadParamMSize + paramInd*InMat.NComps+cSegToComp[segmentInd] ]
#ifdef NKIN_STATES
	MYDTYPE perThreadStateMSize = InMat.NComps*NKIN_STATES;
#define init_state_macro(stateInd,segmentInd) InitStatesM[NeuronID*perThreadStateMSize + stateInd*InMat.NComps+cSegToComp[segmentInd] ]	;
	SUPERILPMACRO(SET_KINETIC_STATE)
#endif 

	for (int count = 1; count < NILP + 1; count++) {
	if(cBoolModel[PIdx[count] +0*NSEG]){CuInitModel_Ca_HVA(v[count],state_macro(0,count) ,state_macro(1,count) ,param_macro(0, PIdx[count]) , ica[count] ,eca[count] ,cai[count] );}if(cBoolModel[PIdx[count] +1*NSEG]){CuInitModel_Ca_LVAst(v[count],state_macro(2,count) ,state_macro(3,count) ,param_macro(1, PIdx[count]) , ica[count] ,eca[count] ,cai[count] );}if(cBoolModel[PIdx[count] +2*NSEG]){CuInitModel_CaDynamics_E2(v[count],cai[count]  ,param_macro(2, PIdx[count]) ,param_macro(3, PIdx[count]) ,param_macro(4, PIdx[count]) ,param_macro(5, PIdx[count]) , ica[count] ,eca[count] );}if(cBoolModel[PIdx[count] +3*NSEG]){CuInitModel_Ih(v[count],state_macro(5,count) ,param_macro(6, PIdx[count]) ,param_macro(7, PIdx[count]) );}if(cBoolModel[PIdx[count] +4*NSEG]){CuInitModel_Im(v[count],state_macro(6,count) ,param_macro(8, PIdx[count]) );}if(cBoolModel[PIdx[count] +5*NSEG]){CuInitModel_K_Pst(v[count],state_macro(7,count) ,state_macro(8,count) ,param_macro(9, PIdx[count]) );}if(cBoolModel[PIdx[count] +6*NSEG]){CuInitModel_K_Tst(v[count],state_macro(9,count) ,state_macro(10,count) ,param_macro(10, PIdx[count]) );}if(cBoolModel[PIdx[count] +7*NSEG]){CuInitModel_Nap_Et2(v[count],state_macro(11,count) ,state_macro(12,count) ,param_macro(11, PIdx[count]) );}if(cBoolModel[PIdx[count] +8*NSEG]){CuInitModel_NaTa_t(v[count],state_macro(13,count) ,state_macro(14,count) ,param_macro(12, PIdx[count]) );}if(cBoolModel[PIdx[count] +9*NSEG]){CuInitModel_NaTs2_t(v[count],state_macro(15,count) ,state_macro(16,count) ,param_macro(13, PIdx[count]) );}if(cBoolModel[PIdx[count] +10*NSEG]){CuInitModel_pas(v[count],param_macro(14, PIdx[count]) ,param_macro(15, PIdx[count]) );}if(cBoolModel[PIdx[count] +11*NSEG]){CuInitModel_SK_E2(v[count],state_macro(17,count) ,param_macro(16, PIdx[count]) ,param_macro(17, PIdx[count]) , cai[count] ,eca[count] );}if(cBoolModel[PIdx[count] +12*NSEG]){CuInitModel_SKv3_1(v[count],state_macro(18,count) ,param_macro(18, PIdx[count]) );}
	}
	MYDTYPE stimLoc = stim.loc;
	MYFTYPE stimArea = stim.area;
	MYDTYPE dtCounter = 0;
	MYFTYPE  dt = sim.dt;
	MYFTYPE temp;
	for (int i = 0; i<Nt; i++) {
#ifdef STIMFROMCSV
		dt = stim.durs[i];
#endif STIMFROMCSV

		t += 0.5*dt;

		if ((i % (WARPSIZE) == 0)) {
			if (i>0) {
				for (int recInd = 0; recInd<sim.NRecSites; recInd++) {
					VHotGlobal[NeuronID*(sim.NRecSites*Nt*blockDim.y) + threadIdx.y*Nt*sim.NRecSites + recInd*Nt + (i - WARPSIZE) + PIdx[1]] = SMemVHot[WARPSIZE*recInd + PIdx[1]];
				}
			}

			//amps[PIdx[1]] = stim.amps[threadIdx.y*Nt + i + PIdx[1]];
		}
		for (int recInd = 0; recInd<sim.NRecSites; recInd++) {
			if (sim.RecSites[recInd] % WARPSIZE == threadIdx.x)
				//This is done by all threads why???
				SMemVHot[recInd*WARPSIZE + i % (WARPSIZE)] = v[1];//This is going to be challenging to make it general but possible.

		}
		for (int count = 1; count < NILP + 1; count++) {
			rhs[count] = 0;
			D[count] = 0;
			sumCurrents[count] = 0;
			sumConductivity[count] = 0;
			sumCurrentsDv[count] = 0;
			sumConductivityDv[count] = 0;
			StimCurrent[count] = 0;
			ica[count] = 0;//SERIOUSLY??? check if this is correct does not seem right!!!! the whole point of ica is not to be initialized every time step...
			if (PIdx[count] == stimLoc) {
				StimCurrent[count] = 100 * stim.amps[threadIdx.y*Nt + i] / stimArea;
			}
		}
		for (int count = 1; count < NILP + 1; count++) {
		   if(cBoolModel[PIdx[count] +0*NSEG]){CuBreakpointModel_Ca_HVA(sumCurrentsDv[count] , sumConductivityDv[count] ,v[count] +0.001,state_macro(0,count) ,state_macro(1,count) ,param_macro(0, PIdx[count]) , temp,eca[count] ,cai[count] );}if(cBoolModel[PIdx[count] +1*NSEG]){CuBreakpointModel_Ca_LVAst(sumCurrentsDv[count] , sumConductivityDv[count] ,v[count] +0.001,state_macro(2,count) ,state_macro(3,count) ,param_macro(1, PIdx[count]) , temp,eca[count] ,cai[count] );}if(cBoolModel[PIdx[count] +2*NSEG]){CuBreakpointModel_CaDynamics_E2(sumCurrentsDv[count] , sumConductivityDv[count] ,v[count] +0.001,cai[count]  ,param_macro(2, PIdx[count]) ,param_macro(3, PIdx[count]) ,param_macro(4, PIdx[count]) ,param_macro(5, PIdx[count]) , temp,eca[count] );}if(cBoolModel[PIdx[count] +3*NSEG]){CuBreakpointModel_Ih(sumCurrentsDv[count] , sumConductivityDv[count] ,v[count] +0.001,state_macro(5,count) ,param_macro(6, PIdx[count]) ,param_macro(7, PIdx[count]) );}if(cBoolModel[PIdx[count] +4*NSEG]){CuBreakpointModel_Im(sumCurrentsDv[count] , sumConductivityDv[count] ,v[count] +0.001,state_macro(6,count) ,param_macro(8, PIdx[count]) );}if(cBoolModel[PIdx[count] +5*NSEG]){CuBreakpointModel_K_Pst(sumCurrentsDv[count] , sumConductivityDv[count] ,v[count] +0.001,state_macro(7,count) ,state_macro(8,count) ,param_macro(9, PIdx[count]) );}if(cBoolModel[PIdx[count] +6*NSEG]){CuBreakpointModel_K_Tst(sumCurrentsDv[count] , sumConductivityDv[count] ,v[count] +0.001,state_macro(9,count) ,state_macro(10,count) ,param_macro(10, PIdx[count]) );}if(cBoolModel[PIdx[count] +7*NSEG]){CuBreakpointModel_Nap_Et2(sumCurrentsDv[count] , sumConductivityDv[count] ,v[count] +0.001,state_macro(11,count) ,state_macro(12,count) ,param_macro(11, PIdx[count]) );}if(cBoolModel[PIdx[count] +8*NSEG]){CuBreakpointModel_NaTa_t(sumCurrentsDv[count] , sumConductivityDv[count] ,v[count] +0.001,state_macro(13,count) ,state_macro(14,count) ,param_macro(12, PIdx[count]) );}if(cBoolModel[PIdx[count] +9*NSEG]){CuBreakpointModel_NaTs2_t(sumCurrentsDv[count] , sumConductivityDv[count] ,v[count] +0.001,state_macro(15,count) ,state_macro(16,count) ,param_macro(13, PIdx[count]) );}if(cBoolModel[PIdx[count] +10*NSEG]){CuBreakpointModel_pas(sumCurrentsDv[count] , sumConductivityDv[count] ,v[count] +0.001,param_macro(14, PIdx[count]) ,param_macro(15, PIdx[count]) );}if(cBoolModel[PIdx[count] +11*NSEG]){CuBreakpointModel_SK_E2(sumCurrentsDv[count] , sumConductivityDv[count] ,v[count] +0.001,state_macro(17,count) ,param_macro(16, PIdx[count]) ,param_macro(17, PIdx[count]) , cai[count] ,eca[count] );}if(cBoolModel[PIdx[count] +12*NSEG]){CuBreakpointModel_SKv3_1(sumCurrentsDv[count] , sumConductivityDv[count] ,v[count] +0.001,state_macro(18,count) ,param_macro(18, PIdx[count]) );}
		  if(cBoolModel[PIdx[count] +0*NSEG]){CuBreakpointModel_Ca_HVA(sumCurrents[count] , sumConductivity[count],v[count] ,state_macro(0,count) ,state_macro(1,count) ,param_macro(0, PIdx[count]) , ica[count] ,eca[count] ,cai[count] );}if(cBoolModel[PIdx[count] +1*NSEG]){CuBreakpointModel_Ca_LVAst(sumCurrents[count] , sumConductivity[count],v[count] ,state_macro(2,count) ,state_macro(3,count) ,param_macro(1, PIdx[count]) , ica[count] ,eca[count] ,cai[count] );}if(cBoolModel[PIdx[count] +2*NSEG]){CuBreakpointModel_CaDynamics_E2(sumCurrents[count] , sumConductivity[count],v[count] ,cai[count]  ,param_macro(2, PIdx[count]) ,param_macro(3, PIdx[count]) ,param_macro(4, PIdx[count]) ,param_macro(5, PIdx[count]) , ica[count] ,eca[count] );}if(cBoolModel[PIdx[count] +3*NSEG]){CuBreakpointModel_Ih(sumCurrents[count] , sumConductivity[count],v[count] ,state_macro(5,count) ,param_macro(6, PIdx[count]) ,param_macro(7, PIdx[count]) );}if(cBoolModel[PIdx[count] +4*NSEG]){CuBreakpointModel_Im(sumCurrents[count] , sumConductivity[count],v[count] ,state_macro(6,count) ,param_macro(8, PIdx[count]) );}if(cBoolModel[PIdx[count] +5*NSEG]){CuBreakpointModel_K_Pst(sumCurrents[count] , sumConductivity[count],v[count] ,state_macro(7,count) ,state_macro(8,count) ,param_macro(9, PIdx[count]) );}if(cBoolModel[PIdx[count] +6*NSEG]){CuBreakpointModel_K_Tst(sumCurrents[count] , sumConductivity[count],v[count] ,state_macro(9,count) ,state_macro(10,count) ,param_macro(10, PIdx[count]) );}if(cBoolModel[PIdx[count] +7*NSEG]){CuBreakpointModel_Nap_Et2(sumCurrents[count] , sumConductivity[count],v[count] ,state_macro(11,count) ,state_macro(12,count) ,param_macro(11, PIdx[count]) );}if(cBoolModel[PIdx[count] +8*NSEG]){CuBreakpointModel_NaTa_t(sumCurrents[count] , sumConductivity[count],v[count] ,state_macro(13,count) ,state_macro(14,count) ,param_macro(12, PIdx[count]) );}if(cBoolModel[PIdx[count] +9*NSEG]){CuBreakpointModel_NaTs2_t(sumCurrents[count] , sumConductivity[count],v[count] ,state_macro(15,count) ,state_macro(16,count) ,param_macro(13, PIdx[count]) );}if(cBoolModel[PIdx[count] +10*NSEG]){CuBreakpointModel_pas(sumCurrents[count] , sumConductivity[count],v[count] ,param_macro(14, PIdx[count]) ,param_macro(15, PIdx[count]) );}if(cBoolModel[PIdx[count] +11*NSEG]){CuBreakpointModel_SK_E2(sumCurrents[count] , sumConductivity[count],v[count] ,state_macro(17,count) ,param_macro(16, PIdx[count]) ,param_macro(17, PIdx[count]) , cai[count] ,eca[count] );}if(cBoolModel[PIdx[count] +12*NSEG]){CuBreakpointModel_SKv3_1(sumCurrents[count] , sumConductivity[count],v[count] ,state_macro(18,count) ,param_macro(18, PIdx[count]) );}

			
		}
		for (int count = 1; count < NILP + 1; count++) {
			gModel[count] = (sumCurrentsDv[count] - sumCurrents[count]) / EPS_V;
			rhs[count] = StimCurrent[count] - sumCurrents[count];
			D[count] = gModel[count] + cCm[PIdx[count]] / (dt * 1000);
			D[count] -= cF[InMat.N - PIdx[count] - 1];
			dv[count] += bHP[InMat.N - parentIndex[count] - 1] - bHP[InMat.N - PIdx[count] - 1];

		}
		for (int count = 1; count < NILP + 1; count++) {
			rhs[count] -= cF[InMat.N - PIdx[count] - 1] * dv[count];
			bHP[InMat.N - PIdx[count] - 1] = rhs[count];
			uHP[InMat.N - PIdx[count] - 1] = D[count];
		}
		//__syncthreads();
		for (int count = 1; count < NILP + 1; count++) {
			if (cSonNoVec[PIdx[count]] == 1) {
				bHP[InMat.N - parentIndex[count] - 1] += cE[Eidx[count]] * dv[count];
				uHP[InMat.N - parentIndex[count] - 1] -= cE[Eidx[count]];
			}//WILL THIS WORK???
			if (cSonNoVec[PIdx[count]] == 2) {
				bHP[InMat.N - parentIndex[count] - 1] += cE[Eidx[count]] * dv[count];
				uHP[InMat.N - parentIndex[count] - 1] -= cE[Eidx[count]];
			}
		}
		//__syncthreads();
		BeforeLU(InMat, uHP, bHP, InMat.Depth);
#ifdef BKSUB1
		//__syncthreads();
		BkSub(InMat, PX, PF, uHP, bHP, InMat.LognDepth);
		//__syncthreads();
		for (int count = 1; count < NILP + 1; count++) {
			Vmid[count] = PX[InMat.N - PIdx[count] - 1];
			v[count] += Vmid[count];
		}
#endif

		
#ifdef BKSUB2
#define THISCOMMANDHERE37(VARILP) MYFTYPE vTemp_ ## VARILP=Vs[PIdx_ ## VARILP];
		MYFTYPE vTemp[1] = Vs[PIdx[1]];
		__syncthreads();
		BkSub(InMat, uHP, bHP, Vs, InMat.Depth);
		__syncthreads();
		PX = Vs;

#define THISCOMMANDHERE38(VARILP) Vmid_ ## VARILP =PX[InMat.N-PIdx_ ## VARILP -1]; v_ ## VARILP +=Vmid_ ## VARILP ;
		Vmid[1] = PX[InMat.N - PIdx[1] - 1]; v[1] += Vmid[1];
		__syncthreads();
#define THISCOMMANDHERE39(VARILP) Vs[PIdx_ ## VARILP ]= vTemp_ ## VARILP +Vmid_ ## VARILP ;
		Vs[PIdx[1]] = vTemp[1] + Vmid[1];
#endif		
            t+=0.5*dt;

        

//      if(InMat.boolModel[PIdx_1 +0*InMat.N]){CuDerivModel_ca(dt, v_1,ModelStates_1[0],ModelStates_1[1],p0_1 ,p1_1 ,ModelStates_1[8],ModelStates_1[9]);} if(InMat.boolModel[PIdx_1 +1*InMat.N]){CuDerivModel_cad(dt, v_1,ModelStates_1[2],ModelStates_1[9],ModelStates_1[8]);} if(InMat.boolModel[PIdx_1 +2*InMat.N]){CuDerivModel_kca(dt, v_1,ModelStates_1[3],p2_1 ,p3_1 ,p4_1 ,p5_1 ,ModelStates_1[8]);} if(InMat.boolModel[PIdx_1 +3*InMat.N]){CuDerivModel_km(dt, v_1,ModelStates_1[4],p6_1 ,p7_1 ,p8_1 ,p9_1 ,p10_1 );} if(InMat.boolModel[PIdx_1 +4*InMat.N]){CuDerivModel_kv(dt, v_1,ModelStates_1[5],p11_1 ,p12_1 ,p13_1 ,p14_1 ,p15_1 );} if(InMat.boolModel[PIdx_1 +5*InMat.N]){CuDerivModel_na(dt, v_1,ModelStates_1[6],ModelStates_1[7],p16_1 ,p17_1 ,p18_1 ,p19_1 ,p20_1 ,p21_1 ,p22_1 ,p23_1 ,p24_1 ,p25_1 ,p26_1 ,p27_1 );}  if(InMat.boolModel[PIdx_2 +0*InMat.N]){CuDerivModel_ca(dt, v_2,ModelStates_2[0],ModelStates_2[1],p0_2 ,p1_2 ,ModelStates_2[8],ModelStates_2[9]);} if(InMat.boolModel[PIdx_2 +1*InMat.N]){CuDerivModel_cad(dt, v_2,ModelStates_2[2],ModelStates_2[9],ModelStates_2[8]);} if(InMat.boolModel[PIdx_2 +2*InMat.N]){CuDerivModel_kca(dt, v_2,ModelStates_2[3],p2_2 ,p3_2 ,p4_2 ,p5_2 ,ModelStates_2[8]);} if(InMat.boolModel[PIdx_2 +3*InMat.N]){CuDerivModel_km(dt, v_2,ModelStates_2[4],p6_2 ,p7_2 ,p8_2 ,p9_2 ,p10_2 );} if(InMat.boolModel[PIdx_2 +4*InMat.N]){CuDerivModel_kv(dt, v_2,ModelStates_2[5],p11_2 ,p12_2 ,p13_2 ,p14_2 ,p15_2 );} if(InMat.boolModel[PIdx_2 +5*InMat.N]){CuDerivModel_na(dt, v_2,ModelStates_2[6],ModelStates_2[7],p16_2 ,p17_2 ,p18_2 ,p19_2 ,p20_2 ,p21_2 ,p22_2 ,p23_2 ,p24_2 ,p25_2 ,p26_2 ,p27_2 );}  if(InMat.boolModel[PIdx_3 +0*InMat.N]){CuDerivModel_ca(dt, v_3,ModelStates_3[0],ModelStates_3[1],p0_3 ,p1_3 ,ModelStates_3[8],ModelStates_3[9]);} if(InMat.boolModel[PIdx_3 +1*InMat.N]){CuDerivModel_cad(dt, v_3,ModelStates_3[2],ModelStates_3[9],ModelStates_3[8]);} if(InMat.boolModel[PIdx_3 +2*InMat.N]){CuDerivModel_kca(dt, v_3,ModelStates_3[3],p2_3 ,p3_3 ,p4_3 ,p5_3 ,ModelStates_3[8]);} if(InMat.boolModel[PIdx_3 +3*InMat.N]){CuDerivModel_km(dt, v_3,ModelStates_3[4],p6_3 ,p7_3 ,p8_3 ,p9_3 ,p10_3 );} if(InMat.boolModel[PIdx_3 +4*InMat.N]){CuDerivModel_kv(dt, v_3,ModelStates_3[5],p11_3 ,p12_3 ,p13_3 ,p14_3 ,p15_3 );} if(InMat.boolModel[PIdx_3 +5*InMat.N]){CuDerivModel_na(dt, v_3,ModelStates_3[6],ModelStates_3[7],p16_3 ,p17_3 ,p18_3 ,p19_3 ,p20_3 ,p21_3 ,p22_3 ,p23_3 ,p24_3 ,p25_3 ,p26_3 ,p27_3 );}
for (int count = 1; count < NILP + 1; count++) {
	  if(cBoolModel[PIdx[count] +0*NSEG]){CuDerivModel_Ca_HVA(dt, v[count],state_macro(0,count) ,state_macro(1,count) ,param_macro(0, PIdx[count]) , ica[count] );}if(cBoolModel[PIdx[count] +1*NSEG]){CuDerivModel_Ca_LVAst(dt, v[count],state_macro(2,count) ,state_macro(3,count) ,param_macro(1, PIdx[count]) , ica[count] );}if(cBoolModel[PIdx[count] +2*NSEG]){CuDerivModel_CaDynamics_E2(dt, v[count],cai[count]  ,param_macro(2, PIdx[count]) ,param_macro(3, PIdx[count]) ,param_macro(4, PIdx[count]) ,param_macro(5, PIdx[count]) , ica[count] ,eca[count] );}if(cBoolModel[PIdx[count] +3*NSEG]){CuDerivModel_Ih(dt, v[count],state_macro(5,count) ,param_macro(6, PIdx[count]) ,param_macro(7, PIdx[count]) );}if(cBoolModel[PIdx[count] +4*NSEG]){CuDerivModel_Im(dt, v[count],state_macro(6,count) ,param_macro(8, PIdx[count]) );}if(cBoolModel[PIdx[count] +5*NSEG]){CuDerivModel_K_Pst(dt, v[count],state_macro(7,count) ,state_macro(8,count) ,param_macro(9, PIdx[count]) );}if(cBoolModel[PIdx[count] +6*NSEG]){CuDerivModel_K_Tst(dt, v[count],state_macro(9,count) ,state_macro(10,count) ,param_macro(10, PIdx[count]) );}if(cBoolModel[PIdx[count] +7*NSEG]){CuDerivModel_Nap_Et2(dt, v[count],state_macro(11,count) ,state_macro(12,count) ,param_macro(11, PIdx[count]) );}if(cBoolModel[PIdx[count] +8*NSEG]){CuDerivModel_NaTa_t(dt, v[count],state_macro(13,count) ,state_macro(14,count) ,param_macro(12, PIdx[count]) );}if(cBoolModel[PIdx[count] +9*NSEG]){CuDerivModel_NaTs2_t(dt, v[count],state_macro(15,count) ,state_macro(16,count) ,param_macro(13, PIdx[count]) );}if(cBoolModel[PIdx[count] +10*NSEG]){}if(cBoolModel[PIdx[count] +11*NSEG]){CuDerivModel_SK_E2(dt, v[count],state_macro(17,count) ,param_macro(16, PIdx[count]) ,param_macro(17, PIdx[count]) , cai[count] ,eca[count] );}if(cBoolModel[PIdx[count] +12*NSEG]){CuDerivModel_SKv3_1(dt, v[count],state_macro(18,count) ,param_macro(18, PIdx[count]) );}
		}

 
 
    }
	//This one looks suspicious but leaving it and will check it later.
	for (int recInd = 0; recInd<sim.NRecSites; recInd++) {
		VHotGlobal[NeuronID*(sim.NRecSites*Nt*blockDim.y) + threadIdx.y*Nt*sim.NRecSites + recInd*Nt + Nt - WARPSIZE + PIdx[1]] = SMemVHot[WARPSIZE*recInd + PIdx[1]];
	}
}
__global__ void NeuroGPUKernel(Stim stim, MYFTYPE* ParamsM, MYFTYPE* ModelStates, Sim sim, HMat InMat, MYFTYPE *V, MYFTYPE* VHotGlobal, MYDTYPE CompDepth, MYDTYPE CompFDepth)
{

	MYFTYPE *amps, *SMemVHot;
	MYDTYPE offset = 0;
	runSimulation(InMat, ParamsM, ModelStates, V, stim, sim, VHotGlobal);
}
void ReadParamsMatX(const char* FN, MYFTYPE* ParamsM, MYDTYPE NParams, MYDTYPE Nx) {
	char FileName[300];
	sprintf(FileName, "%sForC.mat", FN);
	FILE *fl = fopen(FileName, "rb"); // YYY add FILE*
	if (!fl) {
		printf("Failed to read TreeData.x\n");
		return;
	}
	fread(ParamsM, sizeof(MYFTYPE), Nx*NParams, fl);
	fclose(fl);
}

void ReadParamsMat(const char* FN, MYFTYPE** ParamsM, MYDTYPE NParams, MYDTYPE Nx) {
	char FileName[300];
	//sprintf(FileName,"%s%d.mat",FN,MUL32*32);
	sprintf(FileName, "%sForC.mat", FN);
	FILE *fl = fopen(FileName, "rb"); // YYY add FILE*
	if (!fl) {
		printf("Failed to read TreeData.x\n");
		return;
	}
	for (int i = 0; i<NParams; i++) {
		ParamsM[i] = (MYFTYPE*)malloc(Nx * sizeof(MYFTYPE));
		fread(ParamsM[i], sizeof(MYFTYPE), Nx, fl);
	}
	fclose(fl);
}


void initFrameWork(Stim stim, Sim sim, MYFTYPE* ParamsM, MYFTYPE* InitStatesM, HMat& InMat, MYDTYPE CompDepth, MYDTYPE CompFDepth, MYDTYPE NSets, HMat& Mat_d) {

//	printf("in initframework\n");
	hipError_t cudaStatus;
	int i, j, t;
	// For matrix -
	MYFTYPE *PXOut_d, *PFOut_d;
	MYFTYPE *uHPOut_d, *bHPOut_d;
	Mat_d.N = InMat.N;
	Mat_d.NComps = InMat.NComps;
	Mat_d.Depth = InMat.Depth;
	Mat_d.NModels = InMat.NModels;
	Mat_d.LognDepth = InMat.LognDepth;
	Mat_d.nFathers = InMat.nFathers;
	Mat_d.nCallForFather = InMat.nCallForFather;
	Mat_d.nLRel = InMat.nLRel;
	Mat_d.nFLRel = InMat.nFLRel;
	// 32 data
#ifdef BKSUB1
	//cudaStatus = hipMalloc((void**)&Mat_d.FIdxs, InMat.LognDepth*InMat.N* sizeof(MYDTYPE));
#endif
#ifdef BKSUB2
	cudaStatus = hipMalloc((void**)&Mat_d.KsB, (InMat.N + 1) * sizeof(MYDTYPE));
#endif
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cE), InMat.e, InMat.N * sizeof(MYSECONDFTYPE)));
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cF), InMat.f, InMat.N * sizeof(MYSECONDFTYPE)));
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cKs), InMat.Ks, InMat.N * sizeof(MYDTYPE)));
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cSegToComp), InMat.SegToComp, InMat.N * sizeof(MYDTYPE)));
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cBoolModel), InMat.boolModel, InMat.N * InMat.NModels * sizeof(MYDTYPE)));
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cCm), InMat.Cms, InMat.N * sizeof(MYFTYPE)));
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cSonNoVec), InMat.SonNoVec, InMat.N * sizeof(MYDTYPE)));
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cRelStarts), InMat.RelStarts, InMat.nFathers * sizeof(MYDTYPE)));
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cRelEnds), InMat.RelEnds, InMat.nFathers * sizeof(MYDTYPE)));
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cRelVec), InMat.RelVec, InMat.nCallForFather * sizeof(MYDTYPE)));
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cSegStartI), InMat.SegStartI, (InMat.nCallForFather + 1) * sizeof(MYDTYPE)));
	//CUDA_RT_CALL(hipMemcpy(Mat_d.SegEndI, InMat.SegEndI, (InMat.nCallForFather+1)* sizeof(MYDTYPE), hipMemcpyHostToDevice);
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cSegEndI), InMat.SegEndI, (InMat.nCallForFather + 1) * sizeof(MYDTYPE)));
	//CUDA_RT_CALL(hipMemcpy(Mat_d.Fathers, InMat.Fathers, InMat.nFathers * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cFathers), InMat.Fathers, InMat.nFathers * sizeof(MYDTYPE)));
	// 32 data
#ifdef BKSUB1
	//CUDA_RT_CALL(hipMemcpy(Mat_d.FIdxs, InMat.FIdxs, InMat.LognDepth*InMat.N* sizeof(MYDTYPE), hipMemcpyHostToDevice);
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cFIdxs), InMat.FIdxs, InMat.LognDepth*InMat.N * sizeof(MYDTYPE)));
#endif
#ifdef BKSUB2
	CUDA_RT_CALL(hipMemcpy(Mat_d.KsB, InMat.KsB, (InMat.N + 1) * sizeof(MYDTYPE), hipMemcpyHostToDevice);
#endif
	//CUDA_RT_CALL(hipMemcpy(Mat_d.CompByLevel32, InMat.CompByLevel32, (CompDepth)*WARPSIZE*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cCompByLevel32), InMat.CompByLevel32, (CompDepth)*WARPSIZE * sizeof(MYDTYPE)));
	//CUDA_RT_CALL(hipMemcpy(Mat_d.CompByFLevel32, InMat.CompByFLevel32, (CompFDepth)*WARPSIZE*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cCompByFLevel32), InMat.CompByFLevel32, (CompFDepth)*WARPSIZE * sizeof(MYDTYPE)));


	//CUDA_RT_CALL(hipMemcpy(Mat_d.LRelStarts, InMat.LRelStarts,InMat.nLRel*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cLRelStarts), InMat.LRelStarts, InMat.nLRel * sizeof(MYDTYPE)));
	//CUDA_RT_CALL(hipMemcpy(Mat_d.LRelEnds, InMat.LRelEnds,InMat.nLRel*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cLRelEnds), InMat.LRelEnds, InMat.nLRel * sizeof(MYDTYPE)));
	//CUDA_RT_CALL(hipMemcpy(Mat_d.FLRelStarts, InMat.FLRelStarts,InMat.nFLRel*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cFLRelStarts), InMat.FLRelStarts, InMat.nFLRel * sizeof(MYDTYPE)));
	//CUDA_RT_CALL(hipMemcpy(Mat_d.FLRelEnds, InMat.FLRelEnds,InMat.nFLRel*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cFLRelEnds), InMat.FLRelEnds, InMat.nFLRel * sizeof(MYDTYPE)));
	CUDA_RT_CALL(hipMalloc((void**)&PXOut_d, (InMat.N + 1) * sizeof(MYSECONDFTYPE)));
	CUDA_RT_CALL(hipMalloc((void**)&PFOut_d, (InMat.N + 1) * sizeof(MYSECONDFTYPE)));
	CUDA_RT_CALL(hipDeviceSynchronize());
//	printf("done with all init framework\n");
}





void callKernel(Stim stim, Sim sim, MYFTYPE* ParamsM, MYFTYPE* InitStatesM, HMat& Mat_d, MYFTYPE* V, MYDTYPE CompDepth, MYDTYPE CompFDepth, MYDTYPE prevRuns, MYDTYPE currKernelRun, MYFTYPE* VHotsHost) {
	MYDTYPE Nt = stim.Nt;
	MYFTYPE *d_modelParams, *d_modelStates;
	MYFTYPE *VHotsGlobal;
	MYFTYPE *V_d;
	CUDA_RT_CALL(hipMalloc((void**)&VHotsGlobal, currKernelRun*sim.NRecSites*Nt *stim.NStimuli * sizeof(MYFTYPE)));
	int memSizeForVHotGlobal = Nt*stim.NStimuli*sim.NRecSites;
	MYDTYPE memSizeForModelParams = NPARAMS * Mat_d.NComps;
	MYDTYPE memSizeForInitStatae = NSTATES * Mat_d.NComps;
	CUDA_RT_CALL(hipMalloc((void**)&V_d, Mat_d.N * sizeof(MYFTYPE)));

	CUDA_RT_CALL(hipMemcpy(V_d, V, Mat_d.N * sizeof(MYFTYPE), hipMemcpyHostToDevice));
	Stim stim_d;
	stim_d.NStimuli = stim.NStimuli;
	stim_d.comp = stim.comp;
	stim_d.area = stim.area;
	stim_d.loc = stim.loc;
	stim_d.Nt = stim.Nt;
	Sim sim_d;
	sim_d.Celsius = sim.Celsius;
	sim_d.dt = sim.dt;
	sim_d.NRecSites = sim.NRecSites;
	sim_d.TFinal = sim.TFinal;
#ifdef STIMFROMCSV
	printf("in mallocing loop\n******\n");
	CUDA_RT_CALL(hipMalloc((void**)&stim_d.durs, stim_d.Nt * sizeof(MYFTYPE)));
	CUDA_RT_CALL(hipMalloc((void**)&stim_d.amps, stim_d.Nt*stim.NStimuli * sizeof(MYFTYPE)));
#endif // STIMFROMFILE

	CUDA_RT_CALL(hipMalloc((void**)&sim_d.RecSites, sim_d.NRecSites * sizeof(MYDTYPE)));

#ifdef STIMFROMCSV
	CUDA_RT_CALL(hipMemcpy(stim_d.durs, stim.durs, stim_d.Nt * sizeof(MYFTYPE), hipMemcpyHostToDevice));
	CUDA_RT_CALL(hipMemcpy(stim_d.amps, stim.amps, stim_d.Nt*stim.NStimuli * sizeof(MYFTYPE), hipMemcpyHostToDevice));
#endif // stimf
	CUDA_RT_CALL(hipMemcpy(sim_d.RecSites, sim.RecSites, sim_d.NRecSites * sizeof(MYDTYPE), hipMemcpyHostToDevice));
#ifdef NKIN_STATES
	MYFTYPE *d_initStates;
	CUDA_RT_CALL(hipMalloc((void**)&d_initStates, NSTATES * InMat.NComps *NSets * sizeof(MYFTYPE));
	CUDA_RT_CALL(hipMemcpy(d_initStates, InitStatesM, NSTATES * InMat.NComps * NSets * sizeof(MYFTYPE), hipMemcpyHostToDevice);
#endif 
	CUDA_RT_CALL(hipMalloc((void**)&d_modelParams, NPARAMS * Mat_d.NComps *currKernelRun * sizeof(MYFTYPE)));
	CUDA_RT_CALL(hipMemcpy(d_modelParams, &ParamsM[prevRuns*memSizeForModelParams], NPARAMS * Mat_d.NComps * currKernelRun * sizeof(MYFTYPE), hipMemcpyHostToDevice));
	CUDA_RT_CALL(hipMalloc((void**)&d_modelStates, (NSTATES + 1) * (NSEG) * currKernelRun * sizeof(MYFTYPE)));
	dim3 blockDim(WARPSIZE, stim.NStimuli);
	dim3 gridDim(currKernelRun);


#ifdef NKIN_STATES
	if (streamID == 0) {
		NeuroGPUKernel << <currKernelRun, blockDim, TotalSMem, stream0 >> > (stim_d, &d_modelParams[prevRuns*memSizeForModelParams], &d_initStates[prevRuns*memSizeForInitStatae], sim_d, Mat_d, V_d, &VHotsGlobal[prevRuns*memSizeForVHotGlobal], CompDepth, CompFDepth); // RRR	

		CUDA_RT_CALL(hipMemcpyAsync(&VHotsHost[prevRuns*memSizeForVHotGlobal], &VHotsGlobal[prevRuns*memSizeForVHotGlobal], currKernelRun * Nt * sim.NRecSites * stim.NStimuli * sizeof(MYFTYPE), hipMemcpyDeviceToHost, stream0);
		printf("dev id is %d, cudastatus is %s\n", currDevice, cudaStatus);
	}
#endif
//#ifndef NKIN_STATES
	printf("kernel not ran yet\n");
	NeuroGPUKernel << <currKernelRun, blockDim >> >(stim_d, d_modelParams, d_modelStates, sim_d, Mat_d, V_d, VHotsGlobal, CompDepth, CompFDepth); // RRR	
	printf("kernel ran before memcpyasync currkernel run is %d\n", currKernelRun);
	CUDA_RT_CALL(hipMemcpyAsync(VHotsHost, VHotsGlobal, currKernelRun * Nt * sim.NRecSites * stim.NStimuli * sizeof(MYFTYPE), hipMemcpyDeviceToHost));
	printf("done copying*&*&*&*&*&*&*\n");
}

void stEfork2Main(Stim stim, Sim sim, MYFTYPE* ParamsM, MYFTYPE* InitStatesM, HMat& InMat, MYFTYPE* V, MYDTYPE CompDepth, MYDTYPE CompFDepth, int NSets, int* p2pCapableGPUs, int np2p) {
	MYFTYPE *Vhots;
	MYFTYPE **vhots_dev;
	MYDTYPE Nt = stim.Nt;
	printf("in stefork\n");
	vhots_dev = (MYFTYPE**)(malloc(np2p * sizeof(MYFTYPE*)));
	Vhots = (MYFTYPE*)malloc(NSets*Nt*stim.NStimuli*sim.NRecSites * sizeof(MYFTYPE));
	HMat Mat_d;
	if (np2p == 0) {
		np2p = 1;
	}
	for (int i = 0; i < np2p; i++) {
		printf("calling initframework p2pCapableGPUs[i] is %d\n", p2pCapableGPUs[i]);
		CUDA_RT_CALL(hipSetDevice(p2pCapableGPUs[i]));
	//	hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
		initFrameWork(stim, sim, ParamsM, InitStatesM, InMat, CompDepth, CompFDepth, NSets, Mat_d);
	}
	MYDTYPE prevRuns = 0;
	MYDTYPE currRun;
	if (NSets > np2p) {
		currRun = ceil(NSets / np2p);
	}
	else { currRun = NSets; };

	printf("done initframework dev0 curr Kernel is %d\n", currRun);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	for (int i = 0; i < np2p; i++) {
		if (prevRuns >= NSets)  break;
		CUDA_RT_CALL(hipSetDevice(p2pCapableGPUs[i]));
		printf("calling kernel dev%d\n", p2pCapableGPUs[i]);
		hipHostMalloc((void**)&vhots_dev[i], stim.NStimuli*Nt*sim.NRecSites*currRun * sizeof(MYFTYPE));
		callKernel(stim, sim, ParamsM, InitStatesM, Mat_d, V, CompDepth, CompFDepth, prevRuns, currRun, vhots_dev[i]);
		prevRuns += currRun;

	}
	for (int i = 0; i < np2p; i++) {
		CUDA_RT_CALL(hipSetDevice(p2pCapableGPUs[i]));
		CUDA_RT_CALL(hipDeviceSynchronize());
		printf("done synch%d\n", p2pCapableGPUs[i]);
		if (NSets <np2p) {
			printf("nsets >p2pdevs");
			Vhots = vhots_dev[0];
		}
		else {
			memcpy(&Vhots[(currRun*i)*stim.NStimuli*Nt*sim.NRecSites], vhots_dev[i], stim.NStimuli*Nt*sim.NRecSites*currRun * sizeof(MYFTYPE));
		}
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("it took %f ms\n", milliseconds);
	FILE *file = fopen(TIMES_FN, "w");
	if (file) {
		fprintf(file, "%d,%f\n", NSets, milliseconds);
	}
	else {
		printf("ERR SaveArrayToFile %s\n", TIMES_FN);
	}
	fclose(file);
    int curr_dev;
    CUDA_RT_CALL(hipGetDevice(&curr_dev));
    char FileName[300];
	sprintf(FileName, "%s%d.dat", VHOT_OUT_FN_P,curr_dev);
	SaveArrayToFile(FileName, NSets*Nt*stim.NStimuli*sim.NRecSites, Vhots);
}



