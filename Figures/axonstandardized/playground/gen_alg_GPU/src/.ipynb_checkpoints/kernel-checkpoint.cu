
#include "Util.h"
#include <stdio.h>
#include <stdlib.h>


int main(int argc, char *argv[])
{
	int num_devices;
    //int globalRank;
    int stim_ind;
	hipGetDeviceCount(&num_devices);
	num_devices = 1;
	for (int i = 0; i < num_devices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		//printf("Device Number: %d\n", i);
		//printf("  Device name: %s\n", prop.name);
		//printf("  Memory Clock Rate (KHz): %d\n",		prop.memoryClockRate);
		//printf("  Memory Bus Width (bits): %d\n",	prop.memoryBusWidth);
		//printf("  Peak Memory Bandwidth (GB/s): %f\n\n",		2.0*prop.memoryClockRate*(prop.memoryBusWidth / 8) / 1.0e6);
	}
	//RunByModelSerial();// FOR ROY!!!!!!!!!!!!!!! // Run, output VHot and look
    
    if (argc>1){
        stim_ind = char2int(argv[1]); 
        }
        else{
        stim_ind = 0;
        }
     //if (argc>2){
        //globalRank = char2int(argv[2]); 
        //hipMalloc((void**)&devGlobalRank, sizeof(int));
        //hipMemcpy(devGlobalRank, &globalRank, sizeof(int), hipMemcpyHostToDevice);
        //}
       
    printf("got stim num %d\n", stim_ind);  

	CUDA_RT_CALL(hipSetDevice(stim_ind));
	RunByModelP(argc);
	return 0;
}

