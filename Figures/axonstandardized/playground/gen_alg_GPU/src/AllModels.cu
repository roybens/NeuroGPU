#include "hip/hip_runtime.h"
// Automatically generated CU for E:\GitHub\NeuroGPU\Figures\BBP_fromNeuroGPUMasterBranch./runModel.hoc
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""
#include "AllModels.cuh"

// Universals:
#define PI (3.1415927f)
#define R (8.31441f)
#define FARADAY (96485.309f)
#define ktf (1000.*8.3134*(celsius + 273.15)/FARADAY)
#define DEF_vrest	-65.
#define DEF_nai 10.
#define DEF_nao 140.
#define DEF_ena (115. + DEF_vrest)
#define DEF_ki	54.4
#define DEF_ko 2.5
#define DEF_ek (-12. + DEF_vrest)
#include <math.h>
#define DEF_cai	5.e-5
#define DEF_cao	2.
#define	DEF_eca		12.5 *log(DEF_cao / DEF_cai)

// GGlobals:
#define celsius (34.0)
#define stoprun (0.0)
#define clamp_resist (0.001)
#define secondorder (0.0)

// NGlobals:
// Reversals:
#define ena (50.0f)
#define ek (-85.0f)
#define DEF_eca2 (140.21871199503352f)

// Declarations:
__device__ void Curates_Ca_HVA(MYFTYPE v,MYFTYPE gCa_HVAbar_Ca_HVA,MYFTYPE &hAlpha,MYFTYPE &hBeta,MYFTYPE &hInf,MYFTYPE &hTau,MYFTYPE &mAlpha,MYFTYPE &mBeta,MYFTYPE &mInf,MYFTYPE &mTau);
__device__ void Curates_Ca_LVAst(MYFTYPE v,MYFTYPE gCa_LVAstbar_Ca_LVAst,MYFTYPE &hInf,MYFTYPE &hTau,MYFTYPE &mInf,MYFTYPE &mTau);
__device__ void Curates_Ih(MYFTYPE v,MYFTYPE gIhbar_Ih,MYFTYPE ehcn_Ih,MYFTYPE &mAlpha,MYFTYPE &mBeta,MYFTYPE &mInf,MYFTYPE &mTau);
__device__ void Curates_Im(MYFTYPE v,MYFTYPE gImbar_Im,MYFTYPE &mAlpha,MYFTYPE &mBeta,MYFTYPE &mInf,MYFTYPE &mTau);
__device__ void Curates_K_Pst(MYFTYPE v,MYFTYPE gK_Pstbar_K_Pst,MYFTYPE &hInf,MYFTYPE &hTau,MYFTYPE &mInf,MYFTYPE &mTau);
__device__ void Curates_K_Tst(MYFTYPE v,MYFTYPE gK_Tstbar_K_Tst,MYFTYPE &hInf,MYFTYPE &hTau,MYFTYPE &mInf,MYFTYPE &mTau);
__device__ void Curates_Nap_Et2(MYFTYPE v,MYFTYPE gNap_Et2bar_Nap_Et2,MYFTYPE &hAlpha,MYFTYPE &hBeta,MYFTYPE &hInf,MYFTYPE &hTau,MYFTYPE &mAlpha,MYFTYPE &mBeta,MYFTYPE &mInf,MYFTYPE &mTau);
__device__ void Curates_NaTa_t(MYFTYPE v,MYFTYPE gNaTa_tbar_NaTa_t,MYFTYPE &hAlpha,MYFTYPE &hBeta,MYFTYPE &hInf,MYFTYPE &hTau,MYFTYPE &mAlpha,MYFTYPE &mBeta,MYFTYPE &mInf,MYFTYPE &mTau);
__device__ void Curates_NaTs2_t(MYFTYPE v,MYFTYPE gNaTs2_tbar_NaTs2_t,MYFTYPE &hAlpha,MYFTYPE &hBeta,MYFTYPE &hInf,MYFTYPE &hTau,MYFTYPE &mAlpha,MYFTYPE &mBeta,MYFTYPE &mInf,MYFTYPE &mTau);
__device__ void Curates_SK_E2(MYFTYPE ca,MYFTYPE gSK_E2bar_SK_E2,MYFTYPE zTau_SK_E2,MYFTYPE &zInf);
__device__ void Curates_SKv3_1(MYFTYPE v,MYFTYPE gSKv3_1bar_SKv3_1,MYFTYPE &mInf,MYFTYPE &mTau);
float Cunernst(float ci,float co, float z) {
	if (z == 0) {
		return 0.;
	}
	if (ci <= 0.) {
		return 1e6;
	}else if (co <= 0.) {
		return -1e6;
	}else{
		return ktf/z*log(co/ci);
	}	
}

// Functions:

// Procedures:
__device__ void Curates_Ca_HVA(MYFTYPE v,MYFTYPE gCa_HVAbar_Ca_HVA,MYFTYPE &hAlpha,MYFTYPE &hBeta,MYFTYPE &hInf,MYFTYPE &hTau,MYFTYPE &mAlpha,MYFTYPE &mBeta,MYFTYPE &mInf,MYFTYPE &mTau) {
    if ( ( v  == - 27.0 ) ) {
     v = v + 0.0001 ;
     }
   mAlpha = ( 0.055 * ( - 27.0 - v ) ) / ( exp ( ( - 27.0 - v ) / 3.8 ) - 1.0 ) ;
   mBeta = ( 0.94 * exp ( ( - 75.0 - v ) / 17.0 ) ) ;
   mInf = mAlpha / ( mAlpha + mBeta ) ;
   mTau = 1.0 / ( mAlpha + mBeta ) ;
   hAlpha = ( 0.000457 * exp ( ( - 13.0 - v ) / 50.0 ) ) ;
   hBeta = ( 0.0065 / ( exp ( ( - v - 15.0 ) / 28.0 ) + 1.0 ) ) ;
   hInf = hAlpha / ( hAlpha + hBeta ) ;
   hTau = 1.0 / ( hAlpha + hBeta ) ;
}
__device__ void Curates_Ca_LVAst(MYFTYPE v,MYFTYPE gCa_LVAstbar_Ca_LVAst,MYFTYPE &hInf,MYFTYPE &hTau,MYFTYPE &mInf,MYFTYPE &mTau) {
   MYFTYPE qt ;
 qt = powf( 2.3 , ( ( 34.0 - 21.0 ) / 10.0 ) ) ;
    v = v + 10.0 ;
   mInf = 1.0000 / ( 1.0 + exp ( ( v - - 30.000 ) / - 6.0 ) ) ;
   mTau = ( 5.0000 + 20.0000 / ( 1.0 + exp ( ( v - - 25.000 ) / 5.0 ) ) ) / qt ;
   hInf = 1.0000 / ( 1.0 + exp ( ( v - - 80.000 ) / 6.4 ) ) ;
   hTau = ( 20.0000 + 50.0000 / ( 1.0 + exp ( ( v - - 40.000 ) / 7.0 ) ) ) / qt ;
   v = v - 10.0 ;
}
__device__ void Curates_Ih(MYFTYPE v,MYFTYPE gIhbar_Ih,MYFTYPE ehcn_Ih,MYFTYPE &mAlpha,MYFTYPE &mBeta,MYFTYPE &mInf,MYFTYPE &mTau) {
    if ( v  == - 154.9 ) {
     v = v + 0.0001 ;
     }
   mAlpha = 0.001 * 6.43 * ( v + 154.9 ) / ( exp ( ( v + 154.9 ) / 11.9 ) - 1.0 ) ;
   mBeta = 0.001 * 193.0 * exp ( v / 33.1 ) ;
   mInf = mAlpha / ( mAlpha + mBeta ) ;
   mTau = 1.0 / ( mAlpha + mBeta ) ;
}
__device__ void Curates_Im(MYFTYPE v,MYFTYPE gImbar_Im,MYFTYPE &mAlpha,MYFTYPE &mBeta,MYFTYPE &mInf,MYFTYPE &mTau) {
   MYFTYPE qt ;
 qt = powf( 2.3 , ( ( 34.0 - 21.0 ) / 10.0 ) ) ;
    mAlpha = 3.3e-3 * exp ( 2.5 * 0.04 * ( v - - 35.0 ) ) ;
   mBeta = 3.3e-3 * exp ( - 2.5 * 0.04 * ( v - - 35.0 ) ) ;
   mInf = mAlpha / ( mAlpha + mBeta ) ;
   mTau = ( 1.0 / ( mAlpha + mBeta ) ) / qt ;
}
__device__ void Curates_K_Pst(MYFTYPE v,MYFTYPE gK_Pstbar_K_Pst,MYFTYPE &hInf,MYFTYPE &hTau,MYFTYPE &mInf,MYFTYPE &mTau) {
   MYFTYPE qt ;
 qt = powf( 2.3 , ( ( 34.0 - 21.0 ) / 10.0 ) ) ;
    v = v + 10.0 ;
   mInf = ( 1.0 / ( 1.0 + exp ( - ( v + 1.0 ) / 12.0 ) ) ) ;
   if ( v < - 50.0 ) {
     mTau = ( 1.25 + 175.03 * exp ( - v * - 0.026 ) ) / qt ;
     }
   else {
     mTau = ( ( 1.25 + 13.0 * exp ( - v * 0.026 ) ) ) / qt ;
     }
   hInf = 1.0 / ( 1.0 + exp ( - ( v + 54.0 ) / - 11.0 ) ) ;
   hTau = ( 360.0 + ( 1010.0 + 24.0 * ( v + 55.0 ) ) * exp ( - powf( ( ( v + 75.0 ) / 48.0 ) , 2.0 ) ) ) / qt ;
   v = v - 10.0 ;
}
__device__ void Curates_K_Tst(MYFTYPE v,MYFTYPE gK_Tstbar_K_Tst,MYFTYPE &hInf,MYFTYPE &hTau,MYFTYPE &mInf,MYFTYPE &mTau) {
   MYFTYPE qt ;
 qt = powf( 2.3 , ( ( 34.0 - 21.0 ) / 10.0 ) ) ;
    v = v + 10.0 ;
   mInf = 1.0 / ( 1.0 + exp ( - ( v + 0.0 ) / 19.0 ) ) ;
   mTau = ( 0.34 + 0.92 * exp ( - powf( ( ( v + 71.0 ) / 59.0 ) , 2.0 ) ) ) / qt ;
   hInf = 1.0 / ( 1.0 + exp ( - ( v + 66.0 ) / - 10.0 ) ) ;
   hTau = ( 8.0 + 49.0 * exp ( - powf( ( ( v + 73.0 ) / 23.0 ) , 2.0 ) ) ) / qt ;
   v = v - 10.0 ;
}
__device__ void Curates_Nap_Et2(MYFTYPE v,MYFTYPE gNap_Et2bar_Nap_Et2,MYFTYPE &hAlpha,MYFTYPE &hBeta,MYFTYPE &hInf,MYFTYPE &hTau,MYFTYPE &mAlpha,MYFTYPE &mBeta,MYFTYPE &mInf,MYFTYPE &mTau) {
   MYFTYPE qt ;
 qt = powf( 2.3 , ( ( 34.0 - 21.0 ) / 10.0 ) ) ;
    mInf = 1.0 / ( 1.0 + exp ( ( v - - 52.6 ) / - 4.6 ) ) ;
   if ( v  == - 38.0 ) {
     v = v + 0.0001 ;
     }
   mAlpha = ( 0.182 * ( v - - 38.0 ) ) / ( 1.0 - ( exp ( - ( v - - 38.0 ) / 6.0 ) ) ) ;
   mBeta = ( 0.124 * ( - v - 38.0 ) ) / ( 1.0 - ( exp ( - ( - v - 38.0 ) / 6.0 ) ) ) ;
   mTau = 6.0 * ( 1.0 / ( mAlpha + mBeta ) ) / qt ;
   if ( v  == - 17.0 ) {
     v = v + 0.0001 ;
     }
   if ( v  == - 64.4 ) {
     v = v + 0.0001 ;
     }
   hInf = 1.0 / ( 1.0 + exp ( ( v - - 48.8 ) / 10.0 ) ) ;
   hAlpha = - 2.88e-6 * ( v + 17.0 ) / ( 1.0 - exp ( ( v + 17.0 ) / 4.63 ) ) ;
   hBeta = 6.94e-6 * ( v + 64.4 ) / ( 1.0 - exp ( - ( v + 64.4 ) / 2.63 ) ) ;
   hTau = ( 1.0 / ( hAlpha + hBeta ) ) / qt ;
}
__device__ void Curates_NaTa_t(MYFTYPE v,MYFTYPE gNaTa_tbar_NaTa_t,MYFTYPE &hAlpha,MYFTYPE &hBeta,MYFTYPE &hInf,MYFTYPE &hTau,MYFTYPE &mAlpha,MYFTYPE &mBeta,MYFTYPE &mInf,MYFTYPE &mTau) {
   MYFTYPE qt ;
 qt = powf( 2.3 , ( ( 34.0 - 21.0 ) / 10.0 ) ) ;
    if ( v  == - 38.0 ) {
     v = v + 0.0001 ;
     }
   mAlpha = ( 0.182 * ( v - - 38.0 ) ) / ( 1.0 - ( exp ( - ( v - - 38.0 ) / 6.0 ) ) ) ;
   mBeta = ( 0.124 * ( - v - 38.0 ) ) / ( 1.0 - ( exp ( - ( - v - 38.0 ) / 6.0 ) ) ) ;
   mTau = ( 1.0 / ( mAlpha + mBeta ) ) / qt ;
   mInf = mAlpha / ( mAlpha + mBeta ) ;
   if ( v  == - 66.0 ) {
     v = v + 0.0001 ;
     }
   hAlpha = ( - 0.015 * ( v - - 66.0 ) ) / ( 1.0 - ( exp ( ( v - - 66.0 ) / 6.0 ) ) ) ;
   hBeta = ( - 0.015 * ( - v - 66.0 ) ) / ( 1.0 - ( exp ( ( - v - 66.0 ) / 6.0 ) ) ) ;
   hTau = ( 1.0 / ( hAlpha + hBeta ) ) / qt ;
   hInf = hAlpha / ( hAlpha + hBeta ) ;
}
__device__ void Curates_NaTs2_t(MYFTYPE v,MYFTYPE gNaTs2_tbar_NaTs2_t,MYFTYPE &hAlpha,MYFTYPE &hBeta,MYFTYPE &hInf,MYFTYPE &hTau,MYFTYPE &mAlpha,MYFTYPE &mBeta,MYFTYPE &mInf,MYFTYPE &mTau) {
   MYFTYPE qt ;
 qt = powf( 2.3 , ( ( 34.0 - 21.0 ) / 10.0 ) ) ;
    if ( v  == - 32.0 ) {
     v = v + 0.0001 ;
     }
   mAlpha = ( 0.182 * ( v - - 32.0 ) ) / ( 1.0 - ( exp ( - ( v - - 32.0 ) / 6.0 ) ) ) ;
   mBeta = ( 0.124 * ( - v - 32.0 ) ) / ( 1.0 - ( exp ( - ( - v - 32.0 ) / 6.0 ) ) ) ;
   mInf = mAlpha / ( mAlpha + mBeta ) ;
   mTau = ( 1.0 / ( mAlpha + mBeta ) ) / qt ;
   if ( v  == - 60.0 ) {
     v = v + 0.0001 ;
     }
   hAlpha = ( - 0.015 * ( v - - 60.0 ) ) / ( 1.0 - ( exp ( ( v - - 60.0 ) / 6.0 ) ) ) ;
   hBeta = ( - 0.015 * ( - v - 60.0 ) ) / ( 1.0 - ( exp ( ( - v - 60.0 ) / 6.0 ) ) ) ;
   hInf = hAlpha / ( hAlpha + hBeta ) ;
   hTau = ( 1.0 / ( hAlpha + hBeta ) ) / qt ;
}
__device__ void Curates_SK_E2(MYFTYPE ca,MYFTYPE gSK_E2bar_SK_E2,MYFTYPE zTau_SK_E2,MYFTYPE &zInf) {
   if ( ca < 1e-7 ) {
     ca = ca + 1e-07 ;
     }
   zInf = 1.0 / ( 1.0 + powf( ( 0.00043 / ca ) , 4.8 ) ) ;
}
__device__ void Curates_SKv3_1(MYFTYPE v,MYFTYPE gSKv3_1bar_SKv3_1,MYFTYPE &mInf,MYFTYPE &mTau) {
    mInf = 1.0 / ( 1.0 + exp ( ( ( v - ( 18.700 ) ) / ( - 9.700 ) ) ) ) ;
   mTau = 0.2 * 20.000 / ( 1.0 + exp ( ( ( v - ( - 46.560 ) ) / ( - 44.140 ) ) ) ) ;
}

// Inits:


__device__ void CuInitModel_Ca_HVA(MYFTYPE v,MYFTYPE &m,MYFTYPE &h,MYFTYPE gCa_HVAbar_Ca_HVA, MYFTYPE &ica,MYFTYPE &eca, MYFTYPE &cai){
MYFTYPE hAlpha,hBeta,hInf,hTau,mAlpha,mBeta,mInf,mTau;
eca = ktf/2 *log(DEF_cao / cai);
Curates_Ca_HVA(v,gCa_HVAbar_Ca_HVA,hAlpha,hBeta,hInf,hTau,mAlpha,mBeta,mInf,mTau);
   m = mInf;
   h = hInf;
};


__device__ void CuInitModel_Ca_LVAst(MYFTYPE v,MYFTYPE &m,MYFTYPE &h,MYFTYPE gCa_LVAstbar_Ca_LVAst, MYFTYPE &ica,MYFTYPE &eca, MYFTYPE &cai){
MYFTYPE hInf,hTau,mInf,mTau;
eca = ktf/2 *log(DEF_cao / cai);
Curates_Ca_LVAst(v,gCa_LVAstbar_Ca_LVAst,hInf,hTau,mInf,mTau);
   m = mInf;
   h = hInf;
};


__device__ void CuInitModel_CaDynamics_E2(MYFTYPE v,MYFTYPE &cai,MYFTYPE gamma_CaDynamics_E2,MYFTYPE decay_CaDynamics_E2,MYFTYPE depth_CaDynamics_E2,MYFTYPE minCai_CaDynamics_E2, MYFTYPE ica,MYFTYPE &eca){
cai = DEF_cai;
eca = ktf/2 *log(DEF_cao / cai);
};


__device__ void CuInitModel_Ih(MYFTYPE v,MYFTYPE &m,MYFTYPE gIhbar_Ih,MYFTYPE ehcn_Ih){
MYFTYPE mAlpha,mBeta,mInf,mTau;
Curates_Ih(v,gIhbar_Ih,ehcn_Ih,mAlpha,mBeta,mInf,mTau);
   m = mInf;
};


__device__ void CuInitModel_Im(MYFTYPE v,MYFTYPE &m,MYFTYPE gImbar_Im){
MYFTYPE mAlpha,mBeta,mInf,mTau;
Curates_Im(v,gImbar_Im,mAlpha,mBeta,mInf,mTau);
   m = mInf;
};


__device__ void CuInitModel_K_Pst(MYFTYPE v,MYFTYPE &m,MYFTYPE &h,MYFTYPE gK_Pstbar_K_Pst){
MYFTYPE hInf,hTau,mInf,mTau;
Curates_K_Pst(v,gK_Pstbar_K_Pst,hInf,hTau,mInf,mTau);
   m = mInf;
   h = hInf;
};


__device__ void CuInitModel_K_Tst(MYFTYPE v,MYFTYPE &m,MYFTYPE &h,MYFTYPE gK_Tstbar_K_Tst){
MYFTYPE hInf,hTau,mInf,mTau;
Curates_K_Tst(v,gK_Tstbar_K_Tst,hInf,hTau,mInf,mTau);
   m = mInf;
   h = hInf;
};


__device__ void CuInitModel_Nap_Et2(MYFTYPE v,MYFTYPE &m,MYFTYPE &h,MYFTYPE gNap_Et2bar_Nap_Et2){
MYFTYPE hAlpha,hBeta,hInf,hTau,mAlpha,mBeta,mInf,mTau;
Curates_Nap_Et2(v,gNap_Et2bar_Nap_Et2,hAlpha,hBeta,hInf,hTau,mAlpha,mBeta,mInf,mTau);
   m = mInf;
   h = hInf;
};


__device__ void CuInitModel_NaTa_t(MYFTYPE v,MYFTYPE &m,MYFTYPE &h,MYFTYPE gNaTa_tbar_NaTa_t){
MYFTYPE hAlpha,hBeta,hInf,hTau,mAlpha,mBeta,mInf,mTau;
Curates_NaTa_t(v,gNaTa_tbar_NaTa_t,hAlpha,hBeta,hInf,hTau,mAlpha,mBeta,mInf,mTau);
   m = mInf;
   h = hInf;
};


__device__ void CuInitModel_NaTs2_t(MYFTYPE v,MYFTYPE &m,MYFTYPE &h,MYFTYPE gNaTs2_tbar_NaTs2_t){
MYFTYPE hAlpha,hBeta,hInf,hTau,mAlpha,mBeta,mInf,mTau;
Curates_NaTs2_t(v,gNaTs2_tbar_NaTs2_t,hAlpha,hBeta,hInf,hTau,mAlpha,mBeta,mInf,mTau);
   m = mInf;
   h = hInf;
};


__device__ void CuInitModel_pas(MYFTYPE v,MYFTYPE g_pas,MYFTYPE e_pas){
};


__device__ void CuInitModel_SK_E2(MYFTYPE v,MYFTYPE &z,MYFTYPE gSK_E2bar_SK_E2,MYFTYPE zTau_SK_E2, MYFTYPE cai,MYFTYPE &eca){
MYFTYPE zInf;
eca = ktf/2 *log(DEF_cao / cai);
Curates_SK_E2(cai,gSK_E2bar_SK_E2,zTau_SK_E2,zInf);
        z = zInf;
};


__device__ void CuInitModel_SKv3_1(MYFTYPE v,MYFTYPE &m,MYFTYPE gSKv3_1bar_SKv3_1){
MYFTYPE mInf,mTau;
Curates_SKv3_1(v,gSKv3_1bar_SKv3_1,mInf,mTau);
   m = mInf;
};

// Derivs:
__device__ void CuDerivModel_Ca_HVA(MYFTYPE dt, MYFTYPE v,MYFTYPE &m,MYFTYPE &h,MYFTYPE gCa_HVAbar_Ca_HVA, MYFTYPE &ica){
MYFTYPE gCa;
MYFTYPE hAlpha,hBeta,hInf,hTau,mAlpha,mBeta,mInf,mTau;
Curates_Ca_HVA (v,gCa_HVAbar_Ca_HVA,hAlpha,hBeta,hInf,hTau,mAlpha,mBeta,mInf,mTau);
    m = m + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / mTau)))*(- ( ( ( mInf ) ) / mTau ) / ( ( ( ( - 1.0 ) ) ) / mTau ) - m) ;
    h = h + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / hTau)))*(- ( ( ( hInf ) ) / hTau ) / ( ( ( ( - 1.0 ) ) ) / hTau ) - h) ;
}
__device__ void CuDerivModel_Ca_LVAst(MYFTYPE dt, MYFTYPE v,MYFTYPE &m,MYFTYPE &h,MYFTYPE gCa_LVAstbar_Ca_LVAst, MYFTYPE &ica){
MYFTYPE gCa_LVAst;
MYFTYPE hInf,hTau,mInf,mTau;
Curates_Ca_LVAst (v,gCa_LVAstbar_Ca_LVAst,hInf,hTau,mInf,mTau);
    m = m + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / mTau)))*(- ( ( ( mInf ) ) / mTau ) / ( ( ( ( - 1.0 ) ) ) / mTau ) - m) ;
    h = h + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / hTau)))*(- ( ( ( hInf ) ) / hTau ) / ( ( ( ( - 1.0 ) ) ) / hTau ) - h) ;
}
__device__ void CuDerivModel_CaDynamics_E2(MYFTYPE dt, MYFTYPE v,MYFTYPE &cai,MYFTYPE gamma_CaDynamics_E2,MYFTYPE decay_CaDynamics_E2,MYFTYPE depth_CaDynamics_E2,MYFTYPE minCai_CaDynamics_E2, MYFTYPE ica,MYFTYPE &eca){

    cai = cai + (1. - exp(dt*(( - ( ( 1.0 ) ) / decay_CaDynamics_E2 ))))*(- ( ( - ( 10000.0 ) )*( ( ( ( ica )*( gamma_CaDynamics_E2 ) ) / ( 2.0 * FARADAY * depth_CaDynamics_E2 ) ) ) - ( ( ( - minCai_CaDynamics_E2 ) ) ) / decay_CaDynamics_E2 ) / ( ( - ( ( 1.0 ) ) / decay_CaDynamics_E2 ) ) - cai) ;
eca = ktf/2 *log(DEF_cao / cai);
}
__device__ void CuDerivModel_Ih(MYFTYPE dt, MYFTYPE v,MYFTYPE &m,MYFTYPE gIhbar_Ih,MYFTYPE ehcn_Ih){
MYFTYPE ihcn,gIh;
MYFTYPE mAlpha,mBeta,mInf,mTau;
Curates_Ih (v,gIhbar_Ih,ehcn_Ih,mAlpha,mBeta,mInf,mTau);
    m = m + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / mTau)))*(- ( ( ( mInf ) ) / mTau ) / ( ( ( ( - 1.0 ) ) ) / mTau ) - m) ;
}
__device__ void CuDerivModel_Im(MYFTYPE dt, MYFTYPE v,MYFTYPE &m,MYFTYPE gImbar_Im){
MYFTYPE gIm;
MYFTYPE mAlpha,mBeta,mInf,mTau;
Curates_Im (v,gImbar_Im,mAlpha,mBeta,mInf,mTau);
    m = m + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / mTau)))*(- ( ( ( mInf ) ) / mTau ) / ( ( ( ( - 1.0 ) ) ) / mTau ) - m) ;
}
__device__ void CuDerivModel_K_Pst(MYFTYPE dt, MYFTYPE v,MYFTYPE &m,MYFTYPE &h,MYFTYPE gK_Pstbar_K_Pst){
MYFTYPE gK_Pst;
MYFTYPE hInf,hTau,mInf,mTau;
Curates_K_Pst (v,gK_Pstbar_K_Pst,hInf,hTau,mInf,mTau);
    m = m + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / mTau)))*(- ( ( ( mInf ) ) / mTau ) / ( ( ( ( - 1.0 ) ) ) / mTau ) - m) ;
    h = h + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / hTau)))*(- ( ( ( hInf ) ) / hTau ) / ( ( ( ( - 1.0 ) ) ) / hTau ) - h) ;
}
__device__ void CuDerivModel_K_Tst(MYFTYPE dt, MYFTYPE v,MYFTYPE &m,MYFTYPE &h,MYFTYPE gK_Tstbar_K_Tst){
MYFTYPE gK_Tst;
MYFTYPE hInf,hTau,mInf,mTau;
Curates_K_Tst (v,gK_Tstbar_K_Tst,hInf,hTau,mInf,mTau);
    m = m + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / mTau)))*(- ( ( ( mInf ) ) / mTau ) / ( ( ( ( - 1.0 ) ) ) / mTau ) - m) ;
    h = h + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / hTau)))*(- ( ( ( hInf ) ) / hTau ) / ( ( ( ( - 1.0 ) ) ) / hTau ) - h) ;
}
__device__ void CuDerivModel_Nap_Et2(MYFTYPE dt, MYFTYPE v,MYFTYPE &m,MYFTYPE &h,MYFTYPE gNap_Et2bar_Nap_Et2){
MYFTYPE gNap_Et2;
MYFTYPE hAlpha,hBeta,hInf,hTau,mAlpha,mBeta,mInf,mTau;
Curates_Nap_Et2 (v,gNap_Et2bar_Nap_Et2,hAlpha,hBeta,hInf,hTau,mAlpha,mBeta,mInf,mTau);
    m = m + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / mTau)))*(- ( ( ( mInf ) ) / mTau ) / ( ( ( ( - 1.0 ) ) ) / mTau ) - m) ;
    h = h + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / hTau)))*(- ( ( ( hInf ) ) / hTau ) / ( ( ( ( - 1.0 ) ) ) / hTau ) - h) ;
}
__device__ void CuDerivModel_NaTa_t(MYFTYPE dt, MYFTYPE v,MYFTYPE &m,MYFTYPE &h,MYFTYPE gNaTa_tbar_NaTa_t){
MYFTYPE gNaTa_t;
MYFTYPE hAlpha,hBeta,hInf,hTau,mAlpha,mBeta,mInf,mTau;
Curates_NaTa_t (v,gNaTa_tbar_NaTa_t,hAlpha,hBeta,hInf,hTau,mAlpha,mBeta,mInf,mTau);
    m = m + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / mTau)))*(- ( ( ( mInf ) ) / mTau ) / ( ( ( ( - 1.0 ) ) ) / mTau ) - m) ;
    h = h + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / hTau)))*(- ( ( ( hInf ) ) / hTau ) / ( ( ( ( - 1.0 ) ) ) / hTau ) - h) ;
}
__device__ void CuDerivModel_NaTs2_t(MYFTYPE dt, MYFTYPE v,MYFTYPE &m,MYFTYPE &h,MYFTYPE gNaTs2_tbar_NaTs2_t){
MYFTYPE gNaTs2_t;
MYFTYPE hAlpha,hBeta,hInf,hTau,mAlpha,mBeta,mInf,mTau;
Curates_NaTs2_t (v,gNaTs2_tbar_NaTs2_t,hAlpha,hBeta,hInf,hTau,mAlpha,mBeta,mInf,mTau);
    m = m + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / mTau)))*(- ( ( ( mInf ) ) / mTau ) / ( ( ( ( - 1.0 ) ) ) / mTau ) - m) ;
    h = h + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / hTau)))*(- ( ( ( hInf ) ) / hTau ) / ( ( ( ( - 1.0 ) ) ) / hTau ) - h) ;
}
__device__ void CuDerivModel_SK_E2(MYFTYPE dt, MYFTYPE v,MYFTYPE &z,MYFTYPE gSK_E2bar_SK_E2,MYFTYPE zTau_SK_E2, MYFTYPE cai,MYFTYPE &eca){
MYFTYPE gSK_E2;
MYFTYPE zInf;
Curates_SK_E2 (   cai,gSK_E2bar_SK_E2,zTau_SK_E2,zInf);
    z = z + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / zTau_SK_E2)))*(- ( ( ( zInf ) ) / zTau_SK_E2 ) / ( ( ( ( - 1.0 ) ) ) / zTau_SK_E2 ) - z) ;
eca = ktf/2 *log(DEF_cao / cai);
}
__device__ void CuDerivModel_SKv3_1(MYFTYPE dt, MYFTYPE v,MYFTYPE &m,MYFTYPE gSKv3_1bar_SKv3_1){
MYFTYPE gSKv3_1;
MYFTYPE mInf,mTau;
Curates_SKv3_1 (v,gSKv3_1bar_SKv3_1,mInf,mTau);
    m = m + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / mTau)))*(- ( ( ( mInf ) ) / mTau ) / ( ( ( ( - 1.0 ) ) ) / mTau ) - m) ;
}

// Breaks:


__device__ void CuBreakpointModel_Ca_HVA(MYSECONDFTYPE &sumCurrents, MYFTYPE &sumConductivity, MYFTYPE v,MYFTYPE &m,MYFTYPE &h,MYFTYPE gCa_HVAbar_Ca_HVA, MYFTYPE &ica,MYFTYPE &eca, MYFTYPE &cai) {
MYFTYPE gCa_HVA, gCa;
MYFTYPE ;
MYFTYPE ica_Ca_HVA;

   gCa = gCa_HVAbar_Ca_HVA * m * m * h ;
   ica_Ca_HVA = gCa * ( v - eca ) ;
sumCurrents+= ica_Ca_HVA;

 ica += ica_Ca_HVA;
sumConductivity+= gCa;
};


__device__ void CuBreakpointModel_Ca_LVAst(MYSECONDFTYPE &sumCurrents, MYFTYPE &sumConductivity, MYFTYPE v,MYFTYPE &m,MYFTYPE &h,MYFTYPE gCa_LVAstbar_Ca_LVAst, MYFTYPE &ica,MYFTYPE &eca, MYFTYPE &cai) {
MYFTYPE gca, gCa_LVAst;
MYFTYPE ;
MYFTYPE ica_Ca_LVAst;

   gCa_LVAst = gCa_LVAstbar_Ca_LVAst * m * m * h ;
   ica_Ca_LVAst = gCa_LVAst * ( v - eca ) ;
sumCurrents+= ica_Ca_LVAst;

 ica += ica_Ca_LVAst;
sumConductivity+= gCa_LVAst;
};


__device__ void CuBreakpointModel_CaDynamics_E2(MYSECONDFTYPE &sumCurrents, MYFTYPE &sumConductivity, MYFTYPE v,MYFTYPE &cai,MYFTYPE gamma_CaDynamics_E2,MYFTYPE decay_CaDynamics_E2,MYFTYPE depth_CaDynamics_E2,MYFTYPE minCai_CaDynamics_E2, MYFTYPE ica,MYFTYPE &eca) {
MYFTYPE gca;
MYFTYPE ;
};


__device__ void CuBreakpointModel_Ih(MYSECONDFTYPE &sumCurrents, MYFTYPE &sumConductivity, MYFTYPE v,MYFTYPE &m,MYFTYPE gIhbar_Ih,MYFTYPE ehcn_Ih) {
MYFTYPE ihcn, gIh;
MYFTYPE i;
   gIh = gIhbar_Ih * m ;
   ihcn = gIh * ( v - ehcn_Ih ) ;
i = ihcn;

sumCurrents+= i;
sumConductivity+= gIh;
};


__device__ void CuBreakpointModel_Im(MYSECONDFTYPE &sumCurrents, MYFTYPE &sumConductivity, MYFTYPE v,MYFTYPE &m,MYFTYPE gImbar_Im) {
MYFTYPE gk, gIm, ik;
MYFTYPE ;
   gIm = gImbar_Im * m ;
   ik = gIm * ( v - ek ) ;
sumCurrents+= ik;
sumConductivity+= gIm;
};


__device__ void CuBreakpointModel_K_Pst(MYSECONDFTYPE &sumCurrents, MYFTYPE &sumConductivity, MYFTYPE v,MYFTYPE &m,MYFTYPE &h,MYFTYPE gK_Pstbar_K_Pst) {
MYFTYPE gK_Pst, gk, ik;
MYFTYPE ;
   gK_Pst = gK_Pstbar_K_Pst * m * m * h ;
   ik = gK_Pst * ( v - ek ) ;
sumCurrents+= ik;
sumConductivity+= gK_Pst;
};


__device__ void CuBreakpointModel_K_Tst(MYSECONDFTYPE &sumCurrents, MYFTYPE &sumConductivity, MYFTYPE v,MYFTYPE &m,MYFTYPE &h,MYFTYPE gK_Tstbar_K_Tst) {
MYFTYPE gK_Tst, gk, ik;
MYFTYPE ;
   gK_Tst = gK_Tstbar_K_Tst * powf( m , 4.0 ) * h ;
   ik = gK_Tst * ( v - ek ) ;
sumCurrents+= ik;
sumConductivity+= gK_Tst;
};


__device__ void CuBreakpointModel_Nap_Et2(MYSECONDFTYPE &sumCurrents, MYFTYPE &sumConductivity, MYFTYPE v,MYFTYPE &m,MYFTYPE &h,MYFTYPE gNap_Et2bar_Nap_Et2) {
MYFTYPE ina, gNap_Et2, gna;
MYFTYPE ;
   gNap_Et2 = gNap_Et2bar_Nap_Et2 * m * m * m * h ;
   ina = gNap_Et2 * ( v - ena ) ;
sumCurrents+= ina;
sumConductivity+= gNap_Et2;
};


__device__ void CuBreakpointModel_NaTa_t(MYSECONDFTYPE &sumCurrents, MYFTYPE &sumConductivity, MYFTYPE v,MYFTYPE &m,MYFTYPE &h,MYFTYPE gNaTa_tbar_NaTa_t) {
MYFTYPE gNaTa_t, ina, gna;
MYFTYPE ;
   gNaTa_t = gNaTa_tbar_NaTa_t * m * m * m * h ;
   ina = gNaTa_t * ( v - ena ) ;
sumCurrents+= ina;
sumConductivity+= gNaTa_t;
};


__device__ void CuBreakpointModel_NaTs2_t(MYSECONDFTYPE &sumCurrents, MYFTYPE &sumConductivity, MYFTYPE v,MYFTYPE &m,MYFTYPE &h,MYFTYPE gNaTs2_tbar_NaTs2_t) {
MYFTYPE ina, gNaTs2_t, gna;
MYFTYPE ;
   gNaTs2_t = gNaTs2_tbar_NaTs2_t * m * m * m * h ;
   ina = gNaTs2_t * ( v - ena ) ;
sumCurrents+= ina;
sumConductivity+= gNaTs2_t;
};


__device__ void CuBreakpointModel_pas(MYSECONDFTYPE &sumCurrents, MYFTYPE &sumConductivity, MYFTYPE v,MYFTYPE g_pas,MYFTYPE e_pas) {
MYFTYPE;
MYFTYPE i;
   i = g_pas * ( v - e_pas ) ;
i = i;

sumCurrents+= i;
sumConductivity+= g_pas;
};


__device__ void CuBreakpointModel_SK_E2(MYSECONDFTYPE &sumCurrents, MYFTYPE &sumConductivity, MYFTYPE v,MYFTYPE &z,MYFTYPE gSK_E2bar_SK_E2,MYFTYPE zTau_SK_E2, MYFTYPE cai,MYFTYPE &eca) {
MYFTYPE gk, gca, gSK_E2, ik;
MYFTYPE ;
   gSK_E2 = gSK_E2bar_SK_E2 * z ;
   ik = gSK_E2 * ( v - ek ) ;
sumCurrents+= ik;
sumConductivity+= gSK_E2;
};


__device__ void CuBreakpointModel_SKv3_1(MYSECONDFTYPE &sumCurrents, MYFTYPE &sumConductivity, MYFTYPE v,MYFTYPE &m,MYFTYPE gSKv3_1bar_SKv3_1) {
MYFTYPE gk, gSKv3_1, ik;
MYFTYPE ;
   gSKv3_1 = gSKv3_1bar_SKv3_1 * m ;
   ik = gSKv3_1 * ( v - ek ) ;
sumCurrents+= ik;
sumConductivity+= gSKv3_1;
};
