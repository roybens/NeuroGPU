#include "hip/hip_runtime.h"
// Automatically generated CU for E:\GitHub\NeuroGPU\Figures\FigureS3_passive_mew./runModel.hoc
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""
#include "AllModels.cuh"

// Universals:
#define PI (3.1415927f)
#define R (8.31441f)
#define FARADAY (96485.309f)
#define ktf (1000.*8.3134*(celsius + 273.15)/FARADAY)
#define DEF_vrest	-65.
#define DEF_nai 10.
#define DEF_nao 140.
#define DEF_ena (115. + DEF_vrest)
#define DEF_ki	54.4
#define DEF_ko 2.5
#define DEF_ek (-12. + DEF_vrest)
#include <math.h>
#define DEF_cai	5.e-5
#define DEF_cao	2.
#define	DEF_eca		12.5 *log(DEF_cao / DEF_cai)

// GGlobals:
#define celsius (6.3)
#define stoprun (0.0)
#define clamp_resist (0.001)
#define secondorder (0.0)

// NGlobals:

// Declarations:
float Cunernst(float ci,float co, float z) {
	if (z == 0) {
		return 0.;
	}
	if (ci <= 0.) {
		return 1e6;
	}else if (co <= 0.) {
		return -1e6;
	}else{
		return ktf/z*log(co/ci);
	}	
}

// Functions:

// Procedures:

// Inits:


__device__ void CuInitModel_pas(MYFTYPE v,MYFTYPE g_pas,MYFTYPE e_pas){
};

// Derivs:

// Breaks:


__device__ void CuBreakpointModel_pas(MYSECONDFTYPE &sumCurrents, MYFTYPE &sumConductivity, MYFTYPE v,MYFTYPE g_pas,MYFTYPE e_pas) {
MYFTYPE;
MYFTYPE i;
   i = g_pas * ( v - e_pas ) ;
i = i;

sumCurrents+= i;
sumConductivity+= g_pas;
};
