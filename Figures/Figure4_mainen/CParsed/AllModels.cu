#include "hip/hip_runtime.h"
// Automatically generated CU for C:\Users\Maxwell Chen\Desktop\NeuroGPU\Figures\Figure4_mainen./runModel.hoc
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""
#include "AllModels.cuh"

// Universals:
#define PI (3.1415927f)
#define R (8.31441f)
#define FARADAY (96485.309f)
#define ktf (1000.*8.3134*(celsius + 273.15)/FARADAY)
#define DEF_vrest	-65.
#define DEF_nai 10.
#define DEF_nao 140.
#define DEF_ena (115. + DEF_vrest)
#define DEF_ki	54.4
#define DEF_ko 2.5
#define DEF_ek (-12. + DEF_vrest)
#include <math.h>
#define DEF_cai	5.e-5
#define DEF_cao	2.
#define	DEF_eca		12.5 *log(DEF_cao / DEF_cai)

// GGlobals:
#define celsius (37.0)
#define stoprun (0.0)
#define clamp_resist (0.001)
#define secondorder (0.0)

// NGlobals:
#define q10_ca (2.3)
#define temp_ca (23.0)
#define tadj_ca (3.2093639532679714)
#define vmin_ca (-120.0)
#define vmax_ca (100.0)
#define vshift_ca (0.0)
#define depth_cad (0.09334562733124982)
#define cainf_cad (0.0001)
#define taur_cad (200.0)
#define q10_kca (2.3)
#define temp_kca (23.0)
#define tadj_kca (3.2093639532679714)
#define vmin_kca (-120.0)
#define vmax_kca (100.0)
#define q10_km (2.3)
#define temp_km (23.0)
#define tadj_km (3.2093639532679714)
#define vmin_km (-120.0)
#define vmax_km (100.0)
#define q10_kv (2.3)
#define temp_kv (23.0)
#define tadj_kv (3.2093639532679714)
#define vmin_kv (-120.0)
#define vmax_kv (100.0)
#define q10_na (2.3)
#define temp_na (23.0)
#define tadj_na (3.2093639532679714)
#define vmin_na (-120.0)
#define vmax_na (100.0)
#define vshift_na (-5.0)
// Reversals:
#define ek (-90.0f)
#define DEF_eca2 (140.0f)
#define ena (60.0f)

// Declarations:
__device__ void Cutrates_ca(MYFTYPE v ,MYFTYPE gbar_ca,MYFTYPE cao_ca,MYFTYPE &hinf,MYFTYPE &htau,MYFTYPE &minf,MYFTYPE &mtau);
__device__ void Curates_ca(MYFTYPE vm ,MYFTYPE gbar_ca,MYFTYPE cao_ca,MYFTYPE &hinf,MYFTYPE &htau,MYFTYPE &minf,MYFTYPE &mtau);
__device__ void Curates_kca(MYFTYPE cai,MYFTYPE gbar_kca,MYFTYPE caix_kca,MYFTYPE Ra_kca,MYFTYPE Rb_kca,MYFTYPE &a,MYFTYPE &b,MYFTYPE &ninf,MYFTYPE &ntau);
__device__ void Cutrates_km(MYFTYPE v ,MYFTYPE gbar_km,MYFTYPE tha_km,MYFTYPE qa_km,MYFTYPE Ra_km,MYFTYPE Rb_km,MYFTYPE &a,MYFTYPE &b,MYFTYPE &ninf,MYFTYPE &ntau);
__device__ void Curates_km(MYFTYPE v ,MYFTYPE gbar_km,MYFTYPE tha_km,MYFTYPE qa_km,MYFTYPE Ra_km,MYFTYPE Rb_km,MYFTYPE &a,MYFTYPE &b,MYFTYPE &ninf,MYFTYPE &ntau);
__device__ void Cutrates_kv(MYFTYPE v ,MYFTYPE gbar_kv,MYFTYPE tha_kv,MYFTYPE qa_kv,MYFTYPE Ra_kv,MYFTYPE Rb_kv,MYFTYPE &a,MYFTYPE &b,MYFTYPE &ninf,MYFTYPE &ntau);
__device__ void Curates_kv(MYFTYPE v ,MYFTYPE gbar_kv,MYFTYPE tha_kv,MYFTYPE qa_kv,MYFTYPE Ra_kv,MYFTYPE Rb_kv,MYFTYPE &a,MYFTYPE &b,MYFTYPE &ninf,MYFTYPE &ntau);
__device__ void Cutrates_na(MYFTYPE v,MYFTYPE gbar_na,MYFTYPE tha_na,MYFTYPE qa_na,MYFTYPE Ra_na,MYFTYPE Rb_na,MYFTYPE thi1_na,MYFTYPE thi2_na,MYFTYPE qi_na,MYFTYPE thinf_na,MYFTYPE qinf_na,MYFTYPE Rg_na,MYFTYPE Rd_na,MYFTYPE &hinf,MYFTYPE &htau,MYFTYPE &minf,MYFTYPE &mtau);
__device__ void Curates_na(MYFTYPE vm,MYFTYPE gbar_na,MYFTYPE tha_na,MYFTYPE qa_na,MYFTYPE Ra_na,MYFTYPE Rb_na,MYFTYPE thi1_na,MYFTYPE thi2_na,MYFTYPE qi_na,MYFTYPE thinf_na,MYFTYPE qinf_na,MYFTYPE Rg_na,MYFTYPE Rd_na,MYFTYPE &hinf,MYFTYPE &htau,MYFTYPE &minf,MYFTYPE &mtau);
float Cunernst(float ci,float co, float z) {
	if (z == 0) {
		return 0.;
	}
	if (ci <= 0.) {
		return 1e6;
	}else if (co <= 0.) {
		return -1e6;
	}else{
		return ktf/z*log(co/ci);
	}	
}

// Functions:
__device__ MYFTYPE Cuefun_ca(MYFTYPE z){
	if (fabs(z) < 1e-4) {;
		return  1 - z/2;
	}else{;
		return  z/(exp(z) - 1);
	};
};
__device__ MYFTYPE Cuefun_km(MYFTYPE z){
	if (fabs(z) < 1e-4) {;
		return  1 - z/2;
	}else{;
		return  z/(exp(z) - 1);
	};
};
__device__ MYFTYPE Cuefun_kv(MYFTYPE z){
	if (fabs(z) < 1e-4) {;
		return  1 - z/2;
	}else{;
		return  z/(exp(z) - 1);
	};
};
__device__ MYFTYPE Cutrap0_na(MYFTYPE v,MYFTYPE th,MYFTYPE a,MYFTYPE q){
	if (fabs((v-th)/q) > 1e-6) {;
	        return  a * (v - th) / (1 - exp(-(v - th)/q));
	} else {;
	        return  a * q;
	};
}	;

// Procedures:
__device__ void Cutrates_ca(MYFTYPE v ,MYFTYPE gbar_ca,MYFTYPE cao_ca,MYFTYPE &hinf,MYFTYPE &htau,MYFTYPE &minf,MYFTYPE &mtau) {
   Curates_ca (   v ,gbar_ca,cao_ca,hinf,htau,minf,mtau);
}
__device__ void Curates_ca(MYFTYPE vm ,MYFTYPE gbar_ca,MYFTYPE cao_ca,MYFTYPE &hinf,MYFTYPE &htau,MYFTYPE &minf,MYFTYPE &mtau) {
   MYFTYPE a , b ;
/* removed tadj_ca recalculation */
   a = 0.209 * Cuefun_ca (   - ( 27.0 + vm ) / 3.8 ) ;
   b = 0.94 * exp ( ( - 75.0 - vm ) / 17.0 ) ;
   mtau = 1.0 / tadj_ca / ( a + b ) ;
   minf = a / ( a + b ) ;
   a = 0.000457 * exp ( ( - 13.0 - vm ) / 50.0 ) ;
   b = 0.0065 / ( exp ( ( - vm - 15.0 ) / 28.0 ) + 1.0 ) ;
   htau = 1.0 / tadj_ca / ( a + b ) ;
   hinf = a / ( a + b ) ;
}
__device__ void Curates_kca(MYFTYPE cai,MYFTYPE gbar_kca,MYFTYPE caix_kca,MYFTYPE Ra_kca,MYFTYPE Rb_kca,MYFTYPE &a,MYFTYPE &b,MYFTYPE &ninf,MYFTYPE &ntau) {
   a = Ra_kca * powf( cai , caix_kca ) ;
   b = Rb_kca ;
  /* removed tadj_kca recalculation */
   ntau = 1.0 / tadj_kca / ( a + b ) ;
   ninf = a / ( a + b ) ;
}
__device__ void Cutrates_km(MYFTYPE v ,MYFTYPE gbar_km,MYFTYPE tha_km,MYFTYPE qa_km,MYFTYPE Ra_km,MYFTYPE Rb_km,MYFTYPE &a,MYFTYPE &b,MYFTYPE &ninf,MYFTYPE &ntau) {
   Curates_km (   v ,gbar_km,tha_km,qa_km,Ra_km,Rb_km,a,b,ninf,ntau);
}
__device__ void Curates_km(MYFTYPE v ,MYFTYPE gbar_km,MYFTYPE tha_km,MYFTYPE qa_km,MYFTYPE Ra_km,MYFTYPE Rb_km,MYFTYPE &a,MYFTYPE &b,MYFTYPE &ninf,MYFTYPE &ntau) {
   a = Ra_km * qa_km * Cuefun_km (   - ( v - tha_km ) / qa_km ) ;
   b = Rb_km * qa_km * Cuefun_km (   ( v - tha_km ) / qa_km ) ;
  /* removed tadj_km recalculation */
   ntau = 1.0 / tadj_km / ( a + b ) ;
   ninf = a / ( a + b ) ;
}
__device__ void Cutrates_kv(MYFTYPE v ,MYFTYPE gbar_kv,MYFTYPE tha_kv,MYFTYPE qa_kv,MYFTYPE Ra_kv,MYFTYPE Rb_kv,MYFTYPE &a,MYFTYPE &b,MYFTYPE &ninf,MYFTYPE &ntau) {
   Curates_kv (   v ,gbar_kv,tha_kv,qa_kv,Ra_kv,Rb_kv,a,b,ninf,ntau);
}
__device__ void Curates_kv(MYFTYPE v ,MYFTYPE gbar_kv,MYFTYPE tha_kv,MYFTYPE qa_kv,MYFTYPE Ra_kv,MYFTYPE Rb_kv,MYFTYPE &a,MYFTYPE &b,MYFTYPE &ninf,MYFTYPE &ntau) {
   a = Ra_kv * qa_kv * Cuefun_kv (   - ( v - tha_kv ) / qa_kv ) ;
   b = Rb_kv * qa_kv * Cuefun_kv (   ( v - tha_kv ) / qa_kv ) ;
  /* removed tadj_kv recalculation */
   ntau = 1.0 / tadj_kv / ( a + b ) ;
   ninf = a / ( a + b ) ;
}
__device__ void Cutrates_na(MYFTYPE v,MYFTYPE gbar_na,MYFTYPE tha_na,MYFTYPE qa_na,MYFTYPE Ra_na,MYFTYPE Rb_na,MYFTYPE thi1_na,MYFTYPE thi2_na,MYFTYPE qi_na,MYFTYPE thinf_na,MYFTYPE qinf_na,MYFTYPE Rg_na,MYFTYPE Rd_na,MYFTYPE &hinf,MYFTYPE &htau,MYFTYPE &minf,MYFTYPE &mtau) {
   Curates_na (   v ,gbar_na,tha_na,qa_na,Ra_na,Rb_na,thi1_na,thi2_na,qi_na,thinf_na,qinf_na,Rg_na,Rd_na,hinf,htau,minf,mtau);
}
__device__ void Curates_na(MYFTYPE vm,MYFTYPE gbar_na,MYFTYPE tha_na,MYFTYPE qa_na,MYFTYPE Ra_na,MYFTYPE Rb_na,MYFTYPE thi1_na,MYFTYPE thi2_na,MYFTYPE qi_na,MYFTYPE thinf_na,MYFTYPE qinf_na,MYFTYPE Rg_na,MYFTYPE Rd_na,MYFTYPE &hinf,MYFTYPE &htau,MYFTYPE &minf,MYFTYPE &mtau) {
   MYFTYPE a , b ;
 a = Cutrap0_na (   vm , tha_na , Ra_na , qa_na ) ;
   b = Cutrap0_na (   - vm , - tha_na , Rb_na , qa_na ) ;
  /* removed tadj_na recalculation */
   mtau = 1.0 / tadj_na / ( a + b ) ;
   minf = a / ( a + b ) ;
   a = Cutrap0_na (   vm , thi1_na , Rd_na , qi_na ) ;
   b = Cutrap0_na (   - vm , - thi2_na , Rg_na , qi_na ) ;
   htau = 1.0 / tadj_na / ( a + b ) ;
   hinf = 1.0 / ( 1.0 + exp ( ( vm - thinf_na ) / qinf_na ) ) ;
}

// Inits:


__device__ void CuInitModel_ca(MYFTYPE v,MYFTYPE &m,MYFTYPE &h,MYFTYPE gbar_ca,MYFTYPE cao_ca, MYFTYPE cai, MYFTYPE &ica,MYFTYPE &eca){
MYFTYPE hinf,htau,minf,mtau;
eca = DEF_eca2;
eca =DEF_eca2;
   /* removed tadj_ca recalculation */
Cutrates_ca(v+vshift_ca,gbar_ca,cao_ca,hinf,htau,minf,mtau);
   m = minf;
   h = hinf;
};


__device__ void CuInitModel_cad(MYFTYPE v,MYFTYPE &ca, MYFTYPE ica, MYFTYPE &cai,MYFTYPE &eca){
cai = DEF_cai;
eca = ktf/2 *log(DEF_cao / cai);
   ca = cainf_cad;
   cai = ca;
};


__device__ void CuInitModel_kca(MYFTYPE v,MYFTYPE &n,MYFTYPE gbar_kca,MYFTYPE caix_kca,MYFTYPE Ra_kca,MYFTYPE Rb_kca, MYFTYPE cai,MYFTYPE &eca){
MYFTYPE a,b,ninf,ntau;
eca =DEF_eca2;
Curates_kca(cai,gbar_kca,caix_kca,Ra_kca,Rb_kca,a,b,ninf,ntau);
   n = ninf;
};


__device__ void CuInitModel_km(MYFTYPE v,MYFTYPE &n,MYFTYPE gbar_km,MYFTYPE tha_km,MYFTYPE qa_km,MYFTYPE Ra_km,MYFTYPE Rb_km){
MYFTYPE a,b,ninf,ntau;
   /* removed tadj_km recalculation */
Cutrates_km(v,gbar_km,tha_km,qa_km,Ra_km,Rb_km,a,b,ninf,ntau);
   n = ninf;
};


__device__ void CuInitModel_kv(MYFTYPE v,MYFTYPE &n,MYFTYPE gbar_kv,MYFTYPE tha_kv,MYFTYPE qa_kv,MYFTYPE Ra_kv,MYFTYPE Rb_kv){
MYFTYPE a,b,ninf,ntau;
   /* removed tadj_kv recalculation */
Cutrates_kv(v,gbar_kv,tha_kv,qa_kv,Ra_kv,Rb_kv,a,b,ninf,ntau);
   n = ninf;
};


__device__ void CuInitModel_na(MYFTYPE v,MYFTYPE &m,MYFTYPE &h,MYFTYPE gbar_na,MYFTYPE tha_na,MYFTYPE qa_na,MYFTYPE Ra_na,MYFTYPE Rb_na,MYFTYPE thi1_na,MYFTYPE thi2_na,MYFTYPE qi_na,MYFTYPE thinf_na,MYFTYPE qinf_na,MYFTYPE Rg_na,MYFTYPE Rd_na){
MYFTYPE hinf,htau,minf,mtau;
   /* removed tadj_na recalculation */
Cutrates_na(v+vshift_na,gbar_na,tha_na,qa_na,Ra_na,Rb_na,thi1_na,thi2_na,qi_na,thinf_na,qinf_na,Rg_na,Rd_na,hinf,htau,minf,mtau);
   m = minf;
   h = hinf;
};


__device__ void CuInitModel_pas(MYFTYPE v,MYFTYPE g_pas,MYFTYPE e_pas){
};


__device__ void CuInitModel_pas2(MYFTYPE v,MYFTYPE g_pas2,MYFTYPE e_pas2){
};


__device__ void CuInitModel_pasx(MYFTYPE v,MYFTYPE g_pasx,MYFTYPE e_pasx){
};

// Derivs:
__device__ void CuDerivModel_ca(MYFTYPE dt, MYFTYPE v,MYFTYPE &m,MYFTYPE &h,MYFTYPE gbar_ca,MYFTYPE cao_ca, MYFTYPE cai, MYFTYPE &ica,MYFTYPE &eca){

MYFTYPE hinf,htau,minf,mtau;
Cutrates_ca (   v + vshift_ca,gbar_ca,cao_ca,hinf,htau,minf,mtau);
    m = m + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / mtau)))*(- ( ( ( minf ) ) / mtau ) / ( ( ( ( - 1.0 ) ) ) / mtau ) - m) ;
    h = h + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / htau)))*(- ( ( ( hinf ) ) / htau ) / ( ( ( ( - 1.0 ) ) ) / htau ) - h) ;
eca = DEF_eca2;
}
__device__ void CuDerivModel_cad(MYFTYPE dt, MYFTYPE v,MYFTYPE &ca, MYFTYPE ica, MYFTYPE &cai,MYFTYPE &eca){
MYFTYPE drive_channel;
   drive_channel = - ( 10000.0 ) * ica / ( 2.0 * FARADAY * depth_cad ) ;
   if ( drive_channel <= 0. ) {
     drive_channel = 0. ;
     }
    ca = ca + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / taur_cad)))*(- ( drive_channel + ( ( cainf_cad ) ) / taur_cad ) / ( ( ( ( - 1.0 ) ) ) / taur_cad ) - ca) ;
   cai = ca ;
eca = DEF_eca2;
}
__device__ void CuDerivModel_kca(MYFTYPE dt, MYFTYPE v,MYFTYPE &n,MYFTYPE gbar_kca,MYFTYPE caix_kca,MYFTYPE Ra_kca,MYFTYPE Rb_kca, MYFTYPE cai,MYFTYPE &eca){
MYFTYPE ek;
MYFTYPE a,b,ninf,ntau;
Curates_kca (   cai,gbar_kca,caix_kca,Ra_kca,Rb_kca,a,b,ninf,ntau);
    n = n + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / ntau)))*(- ( ( ( ninf ) ) / ntau ) / ( ( ( ( - 1.0 ) ) ) / ntau ) - n) ;
eca = DEF_eca2;
}
__device__ void CuDerivModel_km(MYFTYPE dt, MYFTYPE v,MYFTYPE &n,MYFTYPE gbar_km,MYFTYPE tha_km,MYFTYPE qa_km,MYFTYPE Ra_km,MYFTYPE Rb_km){

MYFTYPE a,b,ninf,ntau;
Cutrates_km (   v,gbar_km,tha_km,qa_km,Ra_km,Rb_km,a,b,ninf,ntau);
    n = n + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / ntau)))*(- ( ( ( ninf ) ) / ntau ) / ( ( ( ( - 1.0 ) ) ) / ntau ) - n) ;
}
__device__ void CuDerivModel_kv(MYFTYPE dt, MYFTYPE v,MYFTYPE &n,MYFTYPE gbar_kv,MYFTYPE tha_kv,MYFTYPE qa_kv,MYFTYPE Ra_kv,MYFTYPE Rb_kv){

MYFTYPE a,b,ninf,ntau;
Cutrates_kv (   v,gbar_kv,tha_kv,qa_kv,Ra_kv,Rb_kv,a,b,ninf,ntau);
    n = n + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / ntau)))*(- ( ( ( ninf ) ) / ntau ) / ( ( ( ( - 1.0 ) ) ) / ntau ) - n) ;
}
__device__ void CuDerivModel_na(MYFTYPE dt, MYFTYPE v,MYFTYPE &m,MYFTYPE &h,MYFTYPE gbar_na,MYFTYPE tha_na,MYFTYPE qa_na,MYFTYPE Ra_na,MYFTYPE Rb_na,MYFTYPE thi1_na,MYFTYPE thi2_na,MYFTYPE qi_na,MYFTYPE thinf_na,MYFTYPE qinf_na,MYFTYPE Rg_na,MYFTYPE Rd_na){

MYFTYPE hinf,htau,minf,mtau;
Cutrates_na (   v + vshift_na,gbar_na,tha_na,qa_na,Ra_na,Rb_na,thi1_na,thi2_na,qi_na,thinf_na,qinf_na,Rg_na,Rd_na,hinf,htau,minf,mtau);
    m = m + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / mtau)))*(- ( ( ( minf ) ) / mtau ) / ( ( ( ( - 1.0 ) ) ) / mtau ) - m) ;
    h = h + (1. - exp(dt*(( ( ( - 1.0 ) ) ) / htau)))*(- ( ( ( hinf ) ) / htau ) / ( ( ( ( - 1.0 ) ) ) / htau ) - h) ;
}

// Breaks:


__device__ void CuBreakpointModel_ca(MYSECONDFTYPE &sumCurrents, MYFTYPE &sumConductivity, MYFTYPE v,MYFTYPE &m,MYFTYPE &h,MYFTYPE gbar_ca,MYFTYPE cao_ca, MYFTYPE cai, MYFTYPE &ica,MYFTYPE &eca) {
MYFTYPE hinf, htau, gca, minf, mtau;
MYFTYPE ;
MYFTYPE ica_ca;

   gca = tadj_ca * gbar_ca * m * m * h ;
   ica_ca = ( 1e-4 ) * gca * ( v - eca ) ;
sumCurrents+= ica_ca;

 ica += ica_ca;
sumConductivity+= gca;
};


__device__ void CuBreakpointModel_cad(MYSECONDFTYPE &sumCurrents, MYFTYPE &sumConductivity, MYFTYPE v,MYFTYPE &ca, MYFTYPE ica, MYFTYPE &cai,MYFTYPE &eca) {
MYFTYPE gca;
MYFTYPE ;
};


__device__ void CuBreakpointModel_kca(MYSECONDFTYPE &sumCurrents, MYFTYPE &sumConductivity, MYFTYPE v,MYFTYPE &n,MYFTYPE gbar_kca,MYFTYPE caix_kca,MYFTYPE Ra_kca,MYFTYPE Rb_kca, MYFTYPE cai,MYFTYPE &eca) {
MYFTYPE gca, gk, ninf, ntau;
MYFTYPE ik;
   gk = tadj_kca * gbar_kca * n ;
   ik = ( 1e-4 ) * gk * ( v - ek ) ;
sumCurrents+= ik;
sumConductivity+= gk;
};


__device__ void CuBreakpointModel_km(MYSECONDFTYPE &sumCurrents, MYFTYPE &sumConductivity, MYFTYPE v,MYFTYPE &n,MYFTYPE gbar_km,MYFTYPE tha_km,MYFTYPE qa_km,MYFTYPE Ra_km,MYFTYPE Rb_km) {
MYFTYPE gk, ninf, ntau;
MYFTYPE ik;
   gk = tadj_km * gbar_km * n ;
   ik = ( 1e-4 ) * gk * ( v - ek ) ;
sumCurrents+= ik;
sumConductivity+= gk;
};


__device__ void CuBreakpointModel_kv(MYSECONDFTYPE &sumCurrents, MYFTYPE &sumConductivity, MYFTYPE v,MYFTYPE &n,MYFTYPE gbar_kv,MYFTYPE tha_kv,MYFTYPE qa_kv,MYFTYPE Ra_kv,MYFTYPE Rb_kv) {
MYFTYPE gk, ninf, ntau;
MYFTYPE ik;
   gk = tadj_kv * gbar_kv * n ;
   ik = ( 1e-4 ) * gk * ( v - ek ) ;
sumCurrents+= ik;
sumConductivity+= gk;
};


__device__ void CuBreakpointModel_na(MYSECONDFTYPE &sumCurrents, MYFTYPE &sumConductivity, MYFTYPE v,MYFTYPE &m,MYFTYPE &h,MYFTYPE gbar_na,MYFTYPE tha_na,MYFTYPE qa_na,MYFTYPE Ra_na,MYFTYPE Rb_na,MYFTYPE thi1_na,MYFTYPE thi2_na,MYFTYPE qi_na,MYFTYPE thinf_na,MYFTYPE qinf_na,MYFTYPE Rg_na,MYFTYPE Rd_na) {
MYFTYPE hinf, htau, gna, minf, mtau;
MYFTYPE ina;
   gna = tadj_na * gbar_na * m * m * m * h ;
   ina = ( 1e-4 ) * gna * ( v - ena ) ;
sumCurrents+= ina;
sumConductivity+= gna;
};


__device__ void CuBreakpointModel_pas(MYSECONDFTYPE &sumCurrents, MYFTYPE &sumConductivity, MYFTYPE v,MYFTYPE g_pas,MYFTYPE e_pas) {
MYFTYPE;
MYFTYPE i;
   i = g_pas * ( v - e_pas ) ;
i = i;

sumCurrents+= i;
sumConductivity+= g_pas;
};


__device__ void CuBreakpointModel_pas2(MYSECONDFTYPE &sumCurrents, MYFTYPE &sumConductivity, MYFTYPE v,MYFTYPE g_pas2,MYFTYPE e_pas2) {
MYFTYPE;
MYFTYPE i;
   i = g_pas2 * ( v - e_pas2 ) ;
i = i;

sumCurrents+= i;
};


__device__ void CuBreakpointModel_pasx(MYSECONDFTYPE &sumCurrents, MYFTYPE &sumConductivity, MYFTYPE v,MYFTYPE g_pasx,MYFTYPE e_pasx) {
MYFTYPE;
MYFTYPE i;
   i = g_pasx * ( v - e_pasx ) ;
i = i;

sumCurrents+= i;
};
