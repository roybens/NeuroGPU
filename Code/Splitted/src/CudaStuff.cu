#include "hip/hip_runtime.h"
#include "CudaStuff.cuh"
//#include "AllModels.cu"
#include "AllModels.cuh"
#define ILP16
__constant__ MYFTYPE cCm[NSEG];
__constant__ MYSECONDFTYPE cE[NSEG];
__constant__ MYSECONDFTYPE cF[NSEG];
__constant__ MYDTYPE cFIdxs[NSEG*LOG_N_DEPTH];
__constant__ MYDTYPE cKs[NSEG];
__constant__ MYDTYPE cSegToComp[NSEG];
__constant__ MYDTYPE cBoolModel[NSEG * N_MODELS];//One day change this to bool
__constant__ MYDTYPE cRelStarts[N_FATHERS];//nFathers
__constant__ MYDTYPE cRelEnds[N_FATHERS];//nFathers
__constant__ MYDTYPE cFathers[N_FATHERS];//nFathers
__constant__ MYDTYPE cRelVec[N_CALL_FOR_FATHER];//nCallForFather
__constant__ MYDTYPE cSegStartI[N_CALL_FOR_FATHER + 1];//nCallForFather
__constant__ MYDTYPE cSegEndI[N_CALL_FOR_FATHER + 1];//nCallForFather

__constant__ MYDTYPE cCompByLevel32[COMP_DEPTH*WARPSIZE];//CompDepth
__constant__ MYDTYPE cCompByFLevel32[COMP_DEPTH*WARPSIZE];//CompFDepth
__constant__ MYDTYPE cLRelStarts[N_L_REL];//nLRel
__constant__ MYDTYPE cLRelEnds[N_L_REL];//nLRel
__constant__ MYDTYPE cFLRelStarts[N_F_L_REL];//nFLRel
__constant__ MYDTYPE cFLRelEnds[N_F_L_REL];//nFLRel
__constant__ MYDTYPE cSonNoVec[NSEG];//InMat.N 


#ifdef __INTELLISENSE__
void __syncthreads();
#endif

#define d_bhp_macro(segmentInd) d_bHP[NeuronID*(NSEG+2)+ segmentInd]
#define d_uhp_macro(segmentInd) d_uHP[NeuronID*(NSEG+2) + segmentInd]

__global__ void BeforeLU(HMat InMat, MYSECONDFTYPE* d_uHP, MYSECONDFTYPE* d_bHP, MYDTYPE Depth)
{
	MYSECONDFTYPE *uHP, *bHP;
	MYDTYPE NeuronID = blockIdx.x;
	MYDTYPE PerStimulus = ((NSEG + 2) * 2 * sizeof(MYSECONDFTYPE));
	PerStimulus = ceilf(MYSECONDFTYPE(PerStimulus) / sizeof(MYSECONDFTYPE)) * sizeof(MYSECONDFTYPE);
	MYDTYPE offset = PerStimulus*threadIdx.y;
	uHP = (MYSECONDFTYPE*)&smem[offset];
	offset += (NSEG + 2) * sizeof(MYSECONDFTYPE);
	bHP = (MYSECONDFTYPE*)&smem[offset];
	offset += (NSEG + 2) * sizeof(MYSECONDFTYPE);
	offset = ceilf(MYSECONDFTYPE(offset) / sizeof(MYSECONDFTYPE)) * sizeof(MYSECONDFTYPE);
	MYDTYPE currIdx = threadIdx.x;
	MYDTYPE i, j, CurJ, CurB, t, CurLevel, LRelIndex;
	MYDTYPE JumctionI;
	MYDTYPE PIdx[NILP + 1];
	for (int count = 1; count < NILP + 1; count++) {
		PIdx[count] = threadIdx.x + (WARPSIZE*(count - 1));
		bHP[PIdx[count]] = d_bhp_macro(PIdx[count]);
		uHP[PIdx[count]] = d_uhp_macro(PIdx[count]);
	}
	//__syncthreads;

	LRelIndex = cLRelStarts[CurLevel];
	LRelIndex = LRelIndex + cLRelEnds[CurLevel];
	for (CurLevel = 0; CurLevel <= Depth; CurLevel++) {

		for (LRelIndex = cLRelStarts[CurLevel]; LRelIndex <= cLRelEnds[CurLevel]; LRelIndex++) {
			//for(LRelIndex=cLRelStarts[CurLevel];LRelIndex<=InMat.LRelEnds[CurLevel];LRelIndex++){ 
			JumctionI = cCompByLevel32[LRelIndex*WARPSIZE + currIdx] - 1;
			for (i = cSegStartI[JumctionI] - 1; i<cSegEndI[JumctionI]; i++) {
				MYSECONDFTYPE uHPm1 = uHP[i - 1];

				uHP[i] = uHP[i] - cF[i - 1] * (cE[i - 1] / uHPm1); // So far same as paper parallel
				uHPm1 = uHP[i - 1];
				MYSECONDFTYPE bHPm1 = bHP[i - 1];
				bHP[i] = bHP[i] - bHPm1*cE[i - 1] / uHPm1; // bH is y
			}
		}
		if (CurLevel<Depth) {
			for (LRelIndex = cFLRelStarts[CurLevel]; LRelIndex <= cFLRelEnds[CurLevel]; LRelIndex++) {
				CurB = cCompByFLevel32[(LRelIndex)*WARPSIZE + currIdx] - 1;//RB i inserted another  -1 into the index RB 2 i removed the-1 from the curlevel 
				CurJ = cFathers[CurB] - 1;
				MYDTYPE St = cRelStarts[CurB];
				MYDTYPE En = cRelEnds[CurB];
				for (j = St; j <= En; j++) {
					t = cRelVec[j - 1] - 1;
					MYSECONDFTYPE uHPm1 = uHP[t - 1];
					uHP[CurJ] -= cF[t - 1] * (cE[t - 1] / uHPm1);
					uHPm1 = uHP[t - 1];
					MYSECONDFTYPE bHPm1 = bHP[t - 1];
					bHP[CurJ] -= bHPm1*cE[t - 1] / uHPm1;
				}
			}
		}
	}
	//__syncthreads;
	for (int count = 1; count < NILP + 1; count++) {
		d_bhp_macro(PIdx[count]) = bHP[PIdx[count]];
		d_uhp_macro(PIdx[count]) = uHP[PIdx[count]];
	}
}

#ifdef BKSUB1
__global__ void BkSub(HMat InMat, MYSECONDFTYPE* d_uHP, MYSECONDFTYPE* d_bHP,  MYDTYPE LognDepth)
{
	MYDTYPE NeuronID = blockIdx.x;
	// MYDTYPE PIdx_1=threadIdx.x; // MYDTYPE NextID_1;
	MYDTYPE PIdx[NILP + 1];
	MYDTYPE NextID[NILP + 1];
	MYSECONDFTYPE *PX,*PF;
	MYDTYPE PerStimulus = ((NSEG + 2) * 2 * sizeof(MYSECONDFTYPE));
	PerStimulus = ceilf(MYSECONDFTYPE(PerStimulus) / sizeof(MYSECONDFTYPE)) * sizeof(MYSECONDFTYPE);
	MYDTYPE offset = PerStimulus*threadIdx.y;
	PF = (MYSECONDFTYPE*)&smem[offset];
	offset += (NSEG + 2) * sizeof(MYSECONDFTYPE);
	PX = (MYSECONDFTYPE*)&smem[offset];
	offset += (NSEG + 2) * sizeof(MYSECONDFTYPE);
	offset = ceilf(MYSECONDFTYPE(offset) / sizeof(MYSECONDFTYPE)) * sizeof(MYSECONDFTYPE);
	for (int count = 1; count < NILP + 1; count++) {
		PIdx[count] = threadIdx.x + (WARPSIZE*(count - 1));
		PX[PIdx[count]] = d_bhp_macro(PIdx[count]);
		PF[PIdx[count]] = d_uhp_macro(PIdx[count]);

	}
	//__syncthreads;
	for (int count = 1; count < NILP + 1; count++) {
		PX[PIdx[count]] = PX[PIdx[count]] / PF[PIdx[count]];
		PF[PIdx[count]] = -cF[PIdx[count]] / PF[PIdx[count]];
		
	}
	MYFTYPE OldPXj[NILP + 1];
	MYFTYPE OldPXNextID[NILP + 1];
	MYFTYPE OldPFj[NILP + 1];
	MYFTYPE OldPFNextID[NILP + 1];

	//#define Replace MYDTYPE PIdx_ ## VARILP =threadIdx.x+(WARPSIZE*(## VARILP -1)); MYDTYPE NextID_ ## VARILP ;
	MYDTYPE i;
	// PX[PIdx_1]=PX[PIdx_1]/PF[PIdx_1]; // PF[PIdx_1]=-cF[PIdx_1]/PF[PIdx_1];
	//#define Replace  PX[PIdx_ ## VARILP ]=PX[PIdx_ ## VARILP ]/PF[PIdx_ ## VARILP ]; PF[PIdx_ ## VARILP ]=-cF[PIdx_ ## VARILP ]/PF[PIdx_ ## VARILP ];


	PX[InMat.N] = 0;
	PF[InMat.N] = 1;
	for (i = 0; i<LognDepth; i++) {
		// NextID_1=cFIdxs[i*InMat.N+PIdx_1]-1; // MYFTYPE OldPXj_1=PX[PIdx_1]; // MYFTYPE OldPXNextID_1=PX[NextID_1]; // PX[PIdx_1]=OldPXj_1+OldPXNextID_1*PF[PIdx_1];
		//#define REPLACE NextID_ ## VARILP =cFIdxs[i*InMat.N+PIdx_ ## VARILP ]-1; MYFTYPE OldPXj_ ## VARILP =PX[PIdx_ ## VARILP ]; MYFTYPE OldPXNextID_ ## VARILP =PX[NextID_ ## VARILP ]; PX[PIdx_ ## VARILP ]=OldPXj_ ## VARILP +OldPXNextID_ ## VARILP *PF[PIdx_ ## VARILP ];
		for (int count = 1; count < NILP + 1; count++) {
			NextID[count] = cFIdxs[i*InMat.N + PIdx[count]] - 1;
			OldPXj[count] = PX[PIdx[count]];
			OldPXNextID[count] = PX[NextID[count]];
			PX[PIdx[count]] = OldPXj[count] + OldPXNextID[count] * PF[PIdx[count]];
		}
		for (int count = 1; count < NILP + 1; count++) {
			OldPFj[count] = PF[PIdx[count]];
			OldPFNextID[count] = PF[NextID[count]];
			PF[PIdx[count]] = OldPFj[count] * OldPFNextID[count];
		}



		// PX[j]=PX[j]+PX[NextID]*PF[j];
		// PF[j]=PF[j]*PF[NextID];
		// MYFTYPE OldPFj_1=PF[PIdx_1]; // MYFTYPE OldPFNextID_1=PF[NextID_1]; // PF[PIdx_1]=OldPFj_1*OldPFNextID_1;
		//#define TReplaced MYFTYPE OldPFj_ ## VARILP =PF[PIdx_ ## VARILP ]; MYFTYPE OldPFNextID_ ## VARILP =PF[NextID_ ## VARILP ]; PF[PIdx_ ## VARILP ]=OldPFj_ ## VARILP *OldPFNextID_ ## VARILP ;

	}
	for (int count = 1; count < NILP + 1; count++) {
		d_bhp_macro(PIdx[count]) = PX[PIdx[count]];
		d_uhp_macro(PIdx[count]) = PF[PIdx[count]];

	}
}
#endif

#ifdef BKSUB2
__device__ void BkSub(HMat InMat, MYSECONDFTYPE* uHP, MYSECONDFTYPE* bHP, MYSECONDFTYPE* Out, MYDTYPE Depth)
{
	// MYDTYPE PIdx_3=threadIdx.x+(WARPSIZE*2); Out[PIdx_2]=0; // might be useless?
#define THISCOMMANDHEREB1(VARILP) MYDTYPE PIdx_ ## VARILP =threadIdx.x+(WARPSIZE*( ## VARILP -1)); Out[PIdx_ ## VARILP ]=0;
	MYDTYPE PIdx[1] = threadIdx.x + (WARPSIZE*([1] - 1)); Out[PIdx[1]] = 0;

	MYDTYPE j, CurJ, CurB, t;
	MYDTYPE JumctionI;
	short CurLevel, i;

	// get KsB from MATLAB (this comes instead of FIdxsX)
	// KsB=Ks; // do in matlab

	// bPX=zeros(1,N); // might be useless
	// for CurLevel=Depth:-1:0
	MYDTYPE LRelIndex, k;
	MYFTYPE temp;
	for (CurLevel = Depth; CurLevel >= 0; CurLevel--) {
		//     Run all independent set for this level, in parallel
		// for JumctionI=find(Level==CurLevel) % in parallel
		for (LRelIndex = cLRelStarts[CurLevel]; LRelIndex <= cLRelEnds[CurLevel]; LRelIndex++) {
			JumctionI = cCompByLevel32[LRelIndex*WARPSIZE + PIdx_1] - 1;
			// for i=(cSegEndI(JumctionI)):-1:(cSegStartI(JumctionI)-1)
			for (i = cSegEndI[JumctionI] - 1; i >= (cSegStartI[JumctionI] - 2); i--) {
				// k=cKsB(i+1);
				k = cKsB[i + 1];

				// bPX(i)=(bH(i)-bPX(k)*f(i))/uH(i);
				Out[i] = (bHP[i] - Out[k] * cF[i]) / uHP[i];
			}
		}
	}
}
#endif
#define perThreadParamMSize NCOMPS*NPARAMS
#define perBlockStatesSize (NSEG)*(NSTATES + 1) 
#define perBlockCaSize (NSEG)
#define perBlockVSize  (NSEG)
#define param_macro(paramInd,segmentInd) ParamsM[NeuronID*perThreadParamMSize + paramInd*NCOMPS+cSegToComp[segmentInd] ]
#define state_macro(stateind,segmentInd) ModelStates[NeuronID*perBlockStatesSize + stateind*NSEG+PIdx[segmentInd]]
#define ca_macro(countInd,ca_arr) ca_arr[NeuronID*perBlockCaSize + PIdx[countInd]]
#define v_macro(segmentInd) V[NeuronID*perBlockVSize + segmentInd]
#define dv_macro(segmentInd) dv[NeuronID*perBlockVSize + segmentInd]
		


__global__ void initModels(HMat InMat,MYFTYPE *V, const MYFTYPE* __restrict__ ParamsM, MYFTYPE* ModelStates,MYFTYPE* cai,MYFTYPE* ica, MYFTYPE* eca ) {
	MYDTYPE PIdx[NILP+1];
	MYFTYPE v[NILP + 1];
	MYDTYPE NeuronID = blockIdx.x;

	for (int count = 1; count < NILP + 1; count++) {
		PIdx[count] = threadIdx.x + (WARPSIZE*(count - 1));
		v[count] = v_macro(PIdx[count]);
	if(cBoolModel[PIdx[count] +0*NSEG]){CuInitModel_Ca_HVA(v[count],state_macro(1,count) ,state_macro(2,count) ,param_macro(0, PIdx[count]) , ica[count] ,eca[count] ,cai[count] );}if(cBoolModel[PIdx[count] +1*NSEG]){CuInitModel_Ca_LVAst(v[count],state_macro(3,count) ,state_macro(4,count) ,param_macro(1, PIdx[count]) , ica[count] ,eca[count] ,cai[count] );}if(cBoolModel[PIdx[count] +2*NSEG]){CuInitModel_CaDynamics_E2(v[count],cai[count]  ,param_macro(2, PIdx[count]) ,param_macro(3, PIdx[count]) ,param_macro(4, PIdx[count]) ,param_macro(5, PIdx[count]) , ica[count] ,eca[count] );}if(cBoolModel[PIdx[count] +3*NSEG]){CuInitModel_Ih(v[count],state_macro(5,count) ,param_macro(6, PIdx[count]) ,param_macro(7, PIdx[count]) );}if(cBoolModel[PIdx[count] +4*NSEG]){CuInitModel_Im(v[count],state_macro(6,count) ,param_macro(8, PIdx[count]) );}if(cBoolModel[PIdx[count] +5*NSEG]){CuInitModel_K_Pst(v[count],state_macro(7,count) ,state_macro(8,count) ,param_macro(9, PIdx[count]) );}if(cBoolModel[PIdx[count] +6*NSEG]){CuInitModel_K_Tst(v[count],state_macro(9,count) ,state_macro(10,count) ,param_macro(10, PIdx[count]) );}if(cBoolModel[PIdx[count] +7*NSEG]){CuInitModel_Nap_Et2(v[count],state_macro(11,count) ,state_macro(12,count) ,param_macro(11, PIdx[count]) );}if(cBoolModel[PIdx[count] +8*NSEG]){CuInitModel_NaTa_t(v[count],state_macro(13,count) ,state_macro(14,count) ,param_macro(12, PIdx[count]) );}if(cBoolModel[PIdx[count] +9*NSEG]){CuInitModel_NaTs2_t(v[count],state_macro(15,count) ,state_macro(16,count) ,param_macro(13, PIdx[count]) );}if(cBoolModel[PIdx[count] +10*NSEG]){CuInitModel_pas(v[count],param_macro(14, PIdx[count]) ,param_macro(15, PIdx[count]) );}if(cBoolModel[PIdx[count] +11*NSEG]){CuInitModel_SK_E2(v[count],state_macro(18,count) ,param_macro(16, PIdx[count]) ,param_macro(17, PIdx[count]) , cai[count] ,eca[count] );}if(cBoolModel[PIdx[count] +12*NSEG]){CuInitModel_SKv3_1(v[count],state_macro(17,count) ,param_macro(18, PIdx[count]) );}
	}
}




//currently only doing one point stimulus

__global__ void updateMatrix(const MYFTYPE* __restrict__ ParamsM, MYFTYPE* ModelStates, MYFTYPE* cai, MYFTYPE* ica, MYFTYPE* eca, MYFTYPE* V, MYFTYPE stimAmp,MYDTYPE stimLoc, MYFTYPE stimArea, MYFTYPE dt,MYFTYPE *d_uHP, MYFTYPE *d_bHP,MYFTYPE *dv)
{

	
	MYSECONDFTYPE *uHP, *bHP;
	//MYDTYPE StimID = threadIdx.y;

#ifdef BKSUB2
	MYDTYPE PerStimulus = ((NSEG + 2) * 2 * sizeof(MYSECONDFTYPE)) + (NSEG + 2 * WARPSIZE) * sizeof(MYFTYPE);
#endif
#ifdef BKSUB1

	MYDTYPE PerStimulus = ((NSEG + 2) * 2 * sizeof(MYSECONDFTYPE));
	PerStimulus = ceilf(MYSECONDFTYPE(PerStimulus) / sizeof(MYSECONDFTYPE)) * sizeof(MYSECONDFTYPE);
#endif

	MYDTYPE offset = PerStimulus*threadIdx.y;
	uHP = (MYSECONDFTYPE*)&smem[offset];
	offset += (NSEG + 2) * sizeof(MYSECONDFTYPE);
	bHP = (MYSECONDFTYPE*)&smem[offset];
	offset += (NSEG + 2) * sizeof(MYSECONDFTYPE);
	offset = ceilf(MYSECONDFTYPE(offset) / sizeof(MYSECONDFTYPE)) * sizeof(MYSECONDFTYPE);
	MYDTYPE NeuronID = blockIdx.x;
	//int Nt = stim.Nt;
	MYFTYPE t = 0;
	//MYSECONDFTYPE *PX, *PF;
	//PX = bHP;
	//PF = uHP;

	MYDTYPE PIdx[NILP + 1];
	for (int count = 1; count < NILP + 1; count++) {
		PIdx[count] = threadIdx.x + (WARPSIZE*(count - 1));
	}

	MYFTYPE Vmid[NILP + 1];
	MYFTYPE v[NILP + 1];
	MYSECONDFTYPE sumCurrents[NILP + 1];
	MYSECONDFTYPE sumCurrentsDv[NILP + 1];
	MYFTYPE sumConductivity[NILP + 1];
	MYFTYPE sumConductivityDv[NILP + 1];
	MYDTYPE parentIndex[NILP + 1];
	MYDTYPE Eidx[NILP + 1];
	MYSECONDFTYPE rhs[NILP + 1];
	MYSECONDFTYPE D[NILP + 1];
	MYFTYPE gModel[NILP + 1];
	MYFTYPE StimCurrent[NILP + 1];
	for (int count = 1; count < NILP + 1; count++) {
		v[count] = v_macro(PIdx[count]);
		sumCurrents[count] = 0;
		sumCurrentsDv[count] = 0;
		sumConductivity[count] = 0;
		sumConductivityDv[count] = 0;
		bHP[PIdx[count]] = d_bhp_macro(PIdx[count]);

		//dv[count] = 0;
		Eidx[count] = NSEG - PIdx[count] - 1;
		parentIndex[count] = NSEG - cKs[NSEG - PIdx[count]];
		if (PIdx[count] == 0) {
			parentIndex[count] = 0;
		};
	}
	if (Eidx[1] > NSEG - 1) {
		Eidx[1] = NSEG - 1;
	}
	MYFTYPE temp;
	for (int count = 1; count < NILP + 1; count++) {
		rhs[count] = 0;
		D[count] = 0;
		sumCurrents[count] = 0;
		sumConductivity[count] = 0;
		sumCurrentsDv[count] = 0;
		sumConductivityDv[count] = 0;
		StimCurrent[count] = 0;
		ca_macro(count,ica) = 0;//SERIOUSLY??? check if this is correct does not seem right!!!! the whole point of ica is not to be initialized every time step...
		if (PIdx[count] == stimLoc) {
			StimCurrent[count] = 100 * stimAmp / stimArea;
		}
	}
	for (int count = 1; count < NILP + 1; count++) {
	   if(cBoolModel[PIdx[count] +0*NSEG]){CuBreakpointModel_Ca_HVA(sumCurrentsDv[count] , sumConductivityDv[count] ,v[count] +0.001,state_macro(1,count) ,state_macro(2,count) ,param_macro(0, PIdx[count]) , temp,eca[count] ,cai[count] );}if(cBoolModel[PIdx[count] +1*NSEG]){CuBreakpointModel_Ca_LVAst(sumCurrentsDv[count] , sumConductivityDv[count] ,v[count] +0.001,state_macro(3,count) ,state_macro(4,count) ,param_macro(1, PIdx[count]) , temp,eca[count] ,cai[count] );}if(cBoolModel[PIdx[count] +2*NSEG]){CuBreakpointModel_CaDynamics_E2(sumCurrentsDv[count] , sumConductivityDv[count] ,v[count] +0.001,cai[count]  ,param_macro(2, PIdx[count]) ,param_macro(3, PIdx[count]) ,param_macro(4, PIdx[count]) ,param_macro(5, PIdx[count]) , temp,eca[count] );}if(cBoolModel[PIdx[count] +3*NSEG]){CuBreakpointModel_Ih(sumCurrentsDv[count] , sumConductivityDv[count] ,v[count] +0.001,state_macro(5,count) ,param_macro(6, PIdx[count]) ,param_macro(7, PIdx[count]) );}if(cBoolModel[PIdx[count] +4*NSEG]){CuBreakpointModel_Im(sumCurrentsDv[count] , sumConductivityDv[count] ,v[count] +0.001,state_macro(6,count) ,param_macro(8, PIdx[count]) );}if(cBoolModel[PIdx[count] +5*NSEG]){CuBreakpointModel_K_Pst(sumCurrentsDv[count] , sumConductivityDv[count] ,v[count] +0.001,state_macro(7,count) ,state_macro(8,count) ,param_macro(9, PIdx[count]) );}if(cBoolModel[PIdx[count] +6*NSEG]){CuBreakpointModel_K_Tst(sumCurrentsDv[count] , sumConductivityDv[count] ,v[count] +0.001,state_macro(9,count) ,state_macro(10,count) ,param_macro(10, PIdx[count]) );}if(cBoolModel[PIdx[count] +7*NSEG]){CuBreakpointModel_Nap_Et2(sumCurrentsDv[count] , sumConductivityDv[count] ,v[count] +0.001,state_macro(11,count) ,state_macro(12,count) ,param_macro(11, PIdx[count]) );}if(cBoolModel[PIdx[count] +8*NSEG]){CuBreakpointModel_NaTa_t(sumCurrentsDv[count] , sumConductivityDv[count] ,v[count] +0.001,state_macro(13,count) ,state_macro(14,count) ,param_macro(12, PIdx[count]) );}if(cBoolModel[PIdx[count] +9*NSEG]){CuBreakpointModel_NaTs2_t(sumCurrentsDv[count] , sumConductivityDv[count] ,v[count] +0.001,state_macro(15,count) ,state_macro(16,count) ,param_macro(13, PIdx[count]) );}if(cBoolModel[PIdx[count] +10*NSEG]){CuBreakpointModel_pas(sumCurrentsDv[count] , sumConductivityDv[count] ,v[count] +0.001,param_macro(14, PIdx[count]) ,param_macro(15, PIdx[count]) );}if(cBoolModel[PIdx[count] +11*NSEG]){CuBreakpointModel_SK_E2(sumCurrentsDv[count] , sumConductivityDv[count] ,v[count] +0.001,state_macro(18,count) ,param_macro(16, PIdx[count]) ,param_macro(17, PIdx[count]) , cai[count] ,eca[count] );}if(cBoolModel[PIdx[count] +12*NSEG]){CuBreakpointModel_SKv3_1(sumCurrentsDv[count] , sumConductivityDv[count] ,v[count] +0.001,state_macro(17,count) ,param_macro(18, PIdx[count]) );}
	  if(cBoolModel[PIdx[count] +0*NSEG]){CuBreakpointModel_Ca_HVA(sumCurrents[count] , sumConductivity[count],v[count] ,state_macro(1,count) ,state_macro(2,count) ,param_macro(0, PIdx[count]) , ica[count] ,eca[count] ,cai[count] );}if(cBoolModel[PIdx[count] +1*NSEG]){CuBreakpointModel_Ca_LVAst(sumCurrents[count] , sumConductivity[count],v[count] ,state_macro(3,count) ,state_macro(4,count) ,param_macro(1, PIdx[count]) , ica[count] ,eca[count] ,cai[count] );}if(cBoolModel[PIdx[count] +2*NSEG]){CuBreakpointModel_CaDynamics_E2(sumCurrents[count] , sumConductivity[count],v[count] ,cai[count]  ,param_macro(2, PIdx[count]) ,param_macro(3, PIdx[count]) ,param_macro(4, PIdx[count]) ,param_macro(5, PIdx[count]) , ica[count] ,eca[count] );}if(cBoolModel[PIdx[count] +3*NSEG]){CuBreakpointModel_Ih(sumCurrents[count] , sumConductivity[count],v[count] ,state_macro(5,count) ,param_macro(6, PIdx[count]) ,param_macro(7, PIdx[count]) );}if(cBoolModel[PIdx[count] +4*NSEG]){CuBreakpointModel_Im(sumCurrents[count] , sumConductivity[count],v[count] ,state_macro(6,count) ,param_macro(8, PIdx[count]) );}if(cBoolModel[PIdx[count] +5*NSEG]){CuBreakpointModel_K_Pst(sumCurrents[count] , sumConductivity[count],v[count] ,state_macro(7,count) ,state_macro(8,count) ,param_macro(9, PIdx[count]) );}if(cBoolModel[PIdx[count] +6*NSEG]){CuBreakpointModel_K_Tst(sumCurrents[count] , sumConductivity[count],v[count] ,state_macro(9,count) ,state_macro(10,count) ,param_macro(10, PIdx[count]) );}if(cBoolModel[PIdx[count] +7*NSEG]){CuBreakpointModel_Nap_Et2(sumCurrents[count] , sumConductivity[count],v[count] ,state_macro(11,count) ,state_macro(12,count) ,param_macro(11, PIdx[count]) );}if(cBoolModel[PIdx[count] +8*NSEG]){CuBreakpointModel_NaTa_t(sumCurrents[count] , sumConductivity[count],v[count] ,state_macro(13,count) ,state_macro(14,count) ,param_macro(12, PIdx[count]) );}if(cBoolModel[PIdx[count] +9*NSEG]){CuBreakpointModel_NaTs2_t(sumCurrents[count] , sumConductivity[count],v[count] ,state_macro(15,count) ,state_macro(16,count) ,param_macro(13, PIdx[count]) );}if(cBoolModel[PIdx[count] +10*NSEG]){CuBreakpointModel_pas(sumCurrents[count] , sumConductivity[count],v[count] ,param_macro(14, PIdx[count]) ,param_macro(15, PIdx[count]) );}if(cBoolModel[PIdx[count] +11*NSEG]){CuBreakpointModel_SK_E2(sumCurrents[count] , sumConductivity[count],v[count] ,state_macro(18,count) ,param_macro(16, PIdx[count]) ,param_macro(17, PIdx[count]) , cai[count] ,eca[count] );}if(cBoolModel[PIdx[count] +12*NSEG]){CuBreakpointModel_SKv3_1(sumCurrents[count] , sumConductivity[count],v[count] ,state_macro(17,count) ,param_macro(18, PIdx[count]) );}
	}
	for (int count = 1; count < NILP + 1; count++) {
		gModel[count] = (sumCurrentsDv[count] - sumCurrents[count]) / EPS_V;
		rhs[count] = StimCurrent[count] - sumCurrents[count];
		D[count] = gModel[count] + cCm[PIdx[count]] / (dt * 1000);
		D[count] -= cF[NSEG - PIdx[count] - 1];
		dv_macro(PIdx[count]) += bHP[NSEG - parentIndex[count] - 1] - bHP[NSEG - PIdx[count] - 1];
	}
//	__syncthreads();
	for (int count = 1; count < NILP + 1; count++) {
		rhs[count] -= cF[NSEG - PIdx[count] - 1] * dv_macro(PIdx[count]);
		bHP[NSEG - PIdx[count] - 1] = rhs[count];
		uHP[NSEG - PIdx[count] - 1] = D[count];
	}
//	__syncthreads();
	for (int count = 1; count < NILP + 1; count++) {
		if (cSonNoVec[PIdx[count]] == 1) {
			bHP[NSEG - parentIndex[count] - 1] += cE[Eidx[count]] * dv_macro(PIdx[count]);
			uHP[NSEG - parentIndex[count] - 1] -= cE[Eidx[count]];
		}//WILL THIS WORK???
		if (cSonNoVec[PIdx[count]] == 2) {
			bHP[NSEG - parentIndex[count] - 1] += cE[Eidx[count]] * dv_macro(PIdx[count]);
			uHP[NSEG - parentIndex[count] - 1] -= cE[Eidx[count]];
		}
	}
	//__syncthreads();
	for (int count = 1; count < NILP + 1; count++) {
		d_bhp_macro(PIdx[count]) = bHP[PIdx[count]];
		d_uhp_macro(PIdx[count]) = uHP[PIdx[count]];
	}

}


__global__ void deriv_and_update_v( const MYFTYPE* __restrict__ ParamsM, MYFTYPE* ModelStates, MYFTYPE* cai, MYFTYPE* ica, MYFTYPE* eca,  MYFTYPE *V,MYFTYPE* d_bHP,MYFTYPE dt,MYDTYPE stimLoc,MYFTYPE* Vhot) {
	MYDTYPE PIdx[NILP + 1];
	MYDTYPE NeuronID = blockIdx.x;
	MYDTYPE v[NILP + 1];
	for (int count = 1; count < NILP + 1; count++) {
		PIdx[count] = threadIdx.x + (WARPSIZE*(count - 1));
		v_macro(PIdx[count]) += d_bhp_macro(NSEG - PIdx[count] - 1);
		v[count] = v_macro(PIdx[count]);
	  if(cBoolModel[PIdx[count] +0*NSEG]){CuDerivModel_Ca_HVA(dt, v[count],state_macro(1,count) ,state_macro(2,count) ,param_macro(0, PIdx[count]) , ica[count] );}if(cBoolModel[PIdx[count] +1*NSEG]){CuDerivModel_Ca_LVAst(dt, v[count],state_macro(3,count) ,state_macro(4,count) ,param_macro(1, PIdx[count]) , ica[count] );}if(cBoolModel[PIdx[count] +2*NSEG]){CuDerivModel_CaDynamics_E2(dt, v[count],cai[count]  ,param_macro(2, PIdx[count]) ,param_macro(3, PIdx[count]) ,param_macro(4, PIdx[count]) ,param_macro(5, PIdx[count]) , ica[count] ,eca[count] );}if(cBoolModel[PIdx[count] +3*NSEG]){CuDerivModel_Ih(dt, v[count],state_macro(5,count) ,param_macro(6, PIdx[count]) ,param_macro(7, PIdx[count]) );}if(cBoolModel[PIdx[count] +4*NSEG]){CuDerivModel_Im(dt, v[count],state_macro(6,count) ,param_macro(8, PIdx[count]) );}if(cBoolModel[PIdx[count] +5*NSEG]){CuDerivModel_K_Pst(dt, v[count],state_macro(7,count) ,state_macro(8,count) ,param_macro(9, PIdx[count]) );}if(cBoolModel[PIdx[count] +6*NSEG]){CuDerivModel_K_Tst(dt, v[count],state_macro(9,count) ,state_macro(10,count) ,param_macro(10, PIdx[count]) );}if(cBoolModel[PIdx[count] +7*NSEG]){CuDerivModel_Nap_Et2(dt, v[count],state_macro(11,count) ,state_macro(12,count) ,param_macro(11, PIdx[count]) );}if(cBoolModel[PIdx[count] +8*NSEG]){CuDerivModel_NaTa_t(dt, v[count],state_macro(13,count) ,state_macro(14,count) ,param_macro(12, PIdx[count]) );}if(cBoolModel[PIdx[count] +9*NSEG]){CuDerivModel_NaTs2_t(dt, v[count],state_macro(15,count) ,state_macro(16,count) ,param_macro(13, PIdx[count]) );}if(cBoolModel[PIdx[count] +10*NSEG]){}if(cBoolModel[PIdx[count] +11*NSEG]){CuDerivModel_SK_E2(dt, v[count],state_macro(18,count) ,param_macro(16, PIdx[count]) ,param_macro(17, PIdx[count]) , cai[count] ,eca[count] );}if(cBoolModel[PIdx[count] +12*NSEG]){CuDerivModel_SKv3_1(dt, v[count],state_macro(17,count) ,param_macro(18, PIdx[count]) );}
	}
	if (threadIdx.x == 0) {
		Vhot[NeuronID] = v_macro(stimLoc);
	}

	
}

void ReadParamsMatX(const char* FN, MYFTYPE* ParamsM, MYDTYPE NParams, MYDTYPE Nx) {
	char FileName[300];
	sprintf(FileName, "%sForC.mat", FN);
	FILE *fl = fopen(FileName, "rb"); // YYY add FILE*
	if (!fl) {
		printf("Failed to read TreeData.x\n");
		return;
	}
	fread(ParamsM, sizeof(MYFTYPE), Nx*NParams, fl);
	fclose(fl);
}

void ReadParamsMat(const char* FN, MYFTYPE** ParamsM, MYDTYPE NParams, MYDTYPE Nx) {
	char FileName[300];
	//sprintf(FileName,"%s%d.mat",FN,MUL32*32);
	sprintf(FileName, "%sForC.mat", FN);
	FILE *fl = fopen(FileName, "rb"); // YYY add FILE*
	if (!fl) {
		printf("Failed to read TreeData.x\n");
		return;
	}
	for (int i = 0; i<NParams; i++) {
		ParamsM[i] = (MYFTYPE*)malloc(Nx * sizeof(MYFTYPE));
		fread(ParamsM[i], sizeof(MYFTYPE), Nx, fl);
	}
	fclose(fl);
}


void initFrameWork(Stim stim, Sim sim, MYFTYPE* ParamsM, MYFTYPE* InitStatesM, HMat& InMat, MYDTYPE CompDepth, MYDTYPE CompFDepth, MYDTYPE NSets, HMat& Mat_d) {

	printf("in initframework\n");
	hipError_t cudaStatus;
	int i, j, t;
	// For matrix -
	MYFTYPE *PXOut_d, *PFOut_d;
	MYFTYPE *uHPOut_d, *bHPOut_d;
	Mat_d.N = InMat.N;
	Mat_d.NComps = InMat.NComps;
	Mat_d.Depth = InMat.Depth;
	Mat_d.NModels = InMat.NModels;
	Mat_d.LognDepth = InMat.LognDepth;
	Mat_d.nFathers = InMat.nFathers;
	Mat_d.nCallForFather = InMat.nCallForFather;
	Mat_d.nLRel = InMat.nLRel;
	Mat_d.nFLRel = InMat.nFLRel;
	// 32 data
#ifdef BKSUB1
	//cudaStatus = hipMalloc((void**)&Mat_d.FIdxs, InMat.LognDepth*InMat.N* sizeof(MYDTYPE));
#endif
#ifdef BKSUB2
	cudaStatus = hipMalloc((void**)&Mat_d.KsB, (InMat.N + 1) * sizeof(MYDTYPE));
#endif
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cE), InMat.e, InMat.N * sizeof(MYSECONDFTYPE)));
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cF), InMat.f, InMat.N * sizeof(MYSECONDFTYPE)));
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cKs), InMat.Ks, InMat.N * sizeof(MYDTYPE)));
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cSegToComp), InMat.SegToComp, InMat.N * sizeof(MYDTYPE)));
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cBoolModel), InMat.boolModel, InMat.N * InMat.NModels * sizeof(MYDTYPE)));
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cCm), InMat.Cms, InMat.N * sizeof(MYFTYPE)));
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cSonNoVec), InMat.SonNoVec, InMat.N * sizeof(MYDTYPE)));
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cRelStarts), InMat.RelStarts, InMat.nFathers * sizeof(MYDTYPE)));
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cRelEnds), InMat.RelEnds, InMat.nFathers * sizeof(MYDTYPE)));
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cRelVec), InMat.RelVec, InMat.nCallForFather * sizeof(MYDTYPE)));
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cSegStartI), InMat.SegStartI, (InMat.nCallForFather + 1) * sizeof(MYDTYPE)));
	//CUDA_RT_CALL(hipMemcpy(Mat_d.SegEndI, InMat.SegEndI, (InMat.nCallForFather+1)* sizeof(MYDTYPE), hipMemcpyHostToDevice);
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cSegEndI), InMat.SegEndI, (InMat.nCallForFather + 1) * sizeof(MYDTYPE)));
	//CUDA_RT_CALL(hipMemcpy(Mat_d.Fathers, InMat.Fathers, InMat.nFathers * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cFathers), InMat.Fathers, InMat.nFathers * sizeof(MYDTYPE)));
	// 32 data
#ifdef BKSUB1
	//CUDA_RT_CALL(hipMemcpy(Mat_d.FIdxs, InMat.FIdxs, InMat.LognDepth*InMat.N* sizeof(MYDTYPE), hipMemcpyHostToDevice);
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cFIdxs), InMat.FIdxs, InMat.LognDepth*InMat.N * sizeof(MYDTYPE)));
#endif
#ifdef BKSUB2
	CUDA_RT_CALL(hipMemcpy(Mat_d.KsB, InMat.KsB, (InMat.N + 1) * sizeof(MYDTYPE), hipMemcpyHostToDevice);
#endif
	//CUDA_RT_CALL(hipMemcpy(Mat_d.CompByLevel32, InMat.CompByLevel32, (CompDepth)*WARPSIZE*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cCompByLevel32), InMat.CompByLevel32, (CompDepth)*WARPSIZE * sizeof(MYDTYPE)));
	//CUDA_RT_CALL(hipMemcpy(Mat_d.CompByFLevel32, InMat.CompByFLevel32, (CompFDepth)*WARPSIZE*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cCompByFLevel32), InMat.CompByFLevel32, (CompFDepth)*WARPSIZE * sizeof(MYDTYPE)));


	//CUDA_RT_CALL(hipMemcpy(Mat_d.LRelStarts, InMat.LRelStarts,InMat.nLRel*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cLRelStarts), InMat.LRelStarts, InMat.nLRel * sizeof(MYDTYPE)));
	//CUDA_RT_CALL(hipMemcpy(Mat_d.LRelEnds, InMat.LRelEnds,InMat.nLRel*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cLRelEnds), InMat.LRelEnds, InMat.nLRel * sizeof(MYDTYPE)));
	//CUDA_RT_CALL(hipMemcpy(Mat_d.FLRelStarts, InMat.FLRelStarts,InMat.nFLRel*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cFLRelStarts), InMat.FLRelStarts, InMat.nFLRel * sizeof(MYDTYPE)));
	//CUDA_RT_CALL(hipMemcpy(Mat_d.FLRelEnds, InMat.FLRelEnds,InMat.nFLRel*sizeof(MYDTYPE), hipMemcpyHostToDevice);
	CUDA_RT_CALL(hipMemcpyToSymbol(HIP_SYMBOL(cFLRelEnds), InMat.FLRelEnds, InMat.nFLRel * sizeof(MYDTYPE)));
	CUDA_RT_CALL(hipMalloc((void**)&PXOut_d, (InMat.N + 1) * sizeof(MYSECONDFTYPE)));
	CUDA_RT_CALL(hipMalloc((void**)&PFOut_d, (InMat.N + 1) * sizeof(MYSECONDFTYPE)));
	CUDA_RT_CALL(hipDeviceSynchronize());
	printf("done with all init framework\n");
}





void callKernel(Stim stim, Sim sim, MYFTYPE* ParamsM, MYFTYPE* InitStatesM, HMat& Mat_d, MYFTYPE* V, MYDTYPE CompDepth, MYDTYPE CompFDepth, MYDTYPE prevRuns, MYDTYPE currKernelRun,MYFTYPE* allvsPtr) {
	MYDTYPE Nt = stim.Nt;
	MYFTYPE *d_modelParams, *d_modelStates,*d_cai,*d_ica, *d_eca, *d_uHP,*d_bHP;
	MYFTYPE *V_uva,*dv;
	
	MYDTYPE memSizeForModelParams = NPARAMS * Mat_d.NComps;
	MYDTYPE memSizeForInitStatae = NSTATES * Mat_d.NComps;
	CUDA_RT_CALL(hipMalloc((void**)&V_uva, currKernelRun* Mat_d.N * sizeof(MYFTYPE)));
	for (int i = 0; i < currKernelRun; i++) {
		CUDA_RT_CALL(hipMemcpy(&V_uva[i*Mat_d.N ], V, Mat_d.N * sizeof(MYFTYPE), hipMemcpyHostToDevice));
	}
	CUDA_RT_CALL(hipMallocManaged((void**)&dv, currKernelRun*Mat_d.N * sizeof(MYFTYPE)));
	CUDA_RT_CALL(hipMemset(dv, 0, currKernelRun*Mat_d.N * sizeof(MYFTYPE)));
	Stim stim_d;
	stim_d.NStimuli = stim.NStimuli;
	stim_d.comp = stim.comp;
	stim_d.area = stim.area;
	stim_d.loc = stim.loc;
	stim_d.Nt = stim.Nt;
	Sim sim_d;
	sim_d.Celsius = sim.Celsius;
	sim_d.dt = sim.dt;
	sim_d.NRecSites = sim.NRecSites;
	sim_d.TFinal = sim.TFinal;
#ifndef STIMFROMFILE
	CUDA_RT_CALL(hipMalloc((void**)&stim_d.dels, stim_d.NStimuli * sizeof(MYFTYPE));
	CUDA_RT_CALL(hipMalloc((void**)&stim_d.durs, stim_d.NStimuli * sizeof(MYFTYPE));
	CUDA_RT_CALL(hipMalloc((void**)&stim_d.amps, stim_d.NStimuli * sizeof(MYFTYPE));
#endif // !STIMFROMFILE
#ifndef STIMFROMCSV

	stim_d.numofdts = stim.numofdts;
	CUDA_RT_CALL(hipMalloc((void**)&stim_d.dtInds, stim_d.numofdts * sizeof(MYDTYPE));
	CUDA_RT_CALL(hipMalloc((void**)&stim_d.durs, stim_d.numofdts * sizeof(MYFTYPE));
	CUDA_RT_CALL(hipMalloc((void**)&stim_d.amps, stim_d.Nt*stim.NStimuli * sizeof(MYFTYPE));
#endif // STIMFROMFILE

#ifdef STIMFROMCSV
	printf("in mallocing loop\n******\n");
	CUDA_RT_CALL(hipMalloc((void**)&stim_d.durs, stim_d.Nt * sizeof(MYFTYPE)));
	CUDA_RT_CALL(hipMalloc((void**)&stim_d.amps, stim_d.Nt*stim.NStimuli * sizeof(MYFTYPE)));
#endif // STIMFROMFILE

	CUDA_RT_CALL(hipMalloc((void**)&sim_d.RecSites, sim_d.NRecSites * sizeof(MYDTYPE)));
#ifndef STIMFROMFILE

	CUDA_RT_CALL(hipMemcpy(stim_d.dels, stim.dels, stim.NStimuli * sizeof(MYFTYPE), hipMemcpyHostToDevice);
	CUDA_RT_CALL(hipMemcpy(stim_d.durs, stim.durs, stim.NStimuli * sizeof(MYFTYPE), hipMemcpyHostToDevice);
	CUDA_RT_CALL(hipMemcpy(stim_d.amps, stim.amps, stim.NStimuli * sizeof(MYFTYPE), hipMemcpyHostToDevice);
#endif // !STIMFROMFILE
#ifdef STIMFROMCSV
	CUDA_RT_CALL(hipMemcpy(stim_d.durs, stim.durs, stim_d.Nt * sizeof(MYFTYPE), hipMemcpyHostToDevice));
	CUDA_RT_CALL(hipMemcpy(stim_d.amps, stim.amps, stim_d.Nt*stim.NStimuli * sizeof(MYFTYPE), hipMemcpyHostToDevice));
#endif // stimf
#ifndef STIMFROMCSV
	CUDA_RT_CALL(hipMemcpy(stim_d.dtInds, stim.dtInds, stim.numofdts * sizeof(MYDTYPE), hipMemcpyHostToDevice);
	CUDA_RT_CALL(hipMemcpy(stim_d.durs, stim.durs, stim.numofdts * sizeof(MYFTYPE), hipMemcpyHostToDevice);
	CUDA_RT_CALL(hipMemcpy(stim_d.amps, stim.amps, stim_d.Nt*stim.NStimuli * sizeof(MYFTYPE), hipMemcpyHostToDevice);
#endif // STIMFROMFILE
	CUDA_RT_CALL(hipMemcpy(sim_d.RecSites, sim.RecSites, sim_d.NRecSites * sizeof(MYDTYPE), hipMemcpyHostToDevice));



#ifdef NKIN_STATES
	MYFTYPE *d_initStates;
	CUDA_RT_CALL(hipMalloc((void**)&d_initStates, NSTATES * InMat.NComps *NSets * sizeof(MYFTYPE));
	CUDA_RT_CALL(hipMemcpy(d_initStates, InitStatesM, NSTATES * InMat.NComps * NSets * sizeof(MYFTYPE), hipMemcpyHostToDevice);
#endif 

	CUDA_RT_CALL(hipMalloc((void**)&d_modelParams, NPARAMS * Mat_d.NComps *currKernelRun * sizeof(MYFTYPE)));
	CUDA_RT_CALL(hipMemcpy(d_modelParams, &ParamsM[prevRuns*memSizeForModelParams], NPARAMS * Mat_d.NComps * currKernelRun * sizeof(MYFTYPE), hipMemcpyHostToDevice));
	CUDA_RT_CALL(hipMalloc((void**)&d_modelStates, (NSTATES+1) * (NSEG )* 8*currKernelRun * sizeof(MYFTYPE)));
	CUDA_RT_CALL(hipMalloc((void**)&d_cai, (NSEG)* currKernelRun * sizeof(MYFTYPE)));
	CUDA_RT_CALL(hipMalloc((void**)&d_eca, (NSEG)* currKernelRun * sizeof(MYFTYPE)));
	CUDA_RT_CALL(hipMalloc((void**)&d_ica, (NSEG)* currKernelRun * sizeof(MYFTYPE)));
	CUDA_RT_CALL(hipMemset(d_cai,0, (NSEG)* currKernelRun * sizeof(MYFTYPE)));
	CUDA_RT_CALL(hipMemset(d_ica,0, (NSEG)* currKernelRun * sizeof(MYFTYPE)));
	CUDA_RT_CALL(hipMemset(d_eca,0, (NSEG)* currKernelRun * sizeof(MYFTYPE)));
	CUDA_RT_CALL(hipMalloc((void**)&d_uHP, ((NSEG + 2) *  currKernelRun * sizeof(MYSECONDFTYPE))));
	CUDA_RT_CALL(hipMalloc((void**)&d_bHP, ((NSEG + 2) *  currKernelRun * sizeof(MYSECONDFTYPE))));
	dim3 blockDim(WARPSIZE, stim.NStimuli);
	dim3 gridDim(currKernelRun);
	
	MYDTYPE offset = 0;
#ifdef BKSUB2
	offset += (Mat_d.N + 1) * sizeof(MYDTYPE);
#endif
	MYDTYPE FrameworkMemSize = offset;
	//PerStimulus
	printf("offset is%d\n", offset);

	offset += (Mat_d.N + 2) * sizeof(MYSECONDFTYPE);  // uHP (Diag)
	offset += (Mat_d.N + 2) * sizeof(MYSECONDFTYPE); // bHP (rhs)

	MYDTYPE PerStimulus = offset - FrameworkMemSize;
	MYDTYPE TotalSMem = FrameworkMemSize + (PerStimulus)*stim.NStimuli;
	printf("Asking %d bytes, %d+%d*%d\n", TotalSMem, FrameworkMemSize, PerStimulus, stim.NStimuli);
	initModels << <currKernelRun, blockDim >> > (Mat_d, V_uva, d_modelParams, d_modelStates, d_cai, d_ica, d_eca);
	MYFTYPE stimAmp,dt;
	MYDTYPE stimLoc = stim.loc;
	MYFTYPE stimArea = stim.area;
	MYFTYPE T = 0;
	MYFTYPE* currVsPtr;
	CUDA_RT_CALL(hipMalloc((void**)&currVsPtr, currKernelRun * sizeof(MYSECONDFTYPE)));
	for (int i = 0; i < Nt; i++) {
		stimAmp = stim.amps[i];
		dt = stim.durs[i];
		//hipDeviceSynchronize();
		updateMatrix << <currKernelRun, blockDim, TotalSMem >> > (d_modelParams, d_modelStates, d_cai, d_ica, d_eca, V_uva, stimAmp,stimLoc,stimArea,dt,d_uHP,d_bHP,dv);
		//hipDeviceSynchronize();
		BeforeLU<<<currKernelRun, blockDim, TotalSMem >> >(Mat_d, d_uHP, d_bHP, Mat_d.Depth);
		//hipDeviceSynchronize();
		BkSub << <currKernelRun, blockDim, TotalSMem >> >(Mat_d,d_uHP,d_bHP, Mat_d.LognDepth);
		//hipDeviceSynchronize();
		deriv_and_update_v << <currKernelRun, blockDim>>>(d_modelParams, d_modelStates, d_cai, d_ica, d_eca, V_uva,d_bHP,dt, stim.loc, currVsPtr);
		//hipDeviceSynchronize();
		hipMemcpyAsync(&allvsPtr[i*currKernelRun], currVsPtr, currKernelRun * sizeof(MYFTYPE), hipMemcpyDeviceToHost);
		T += dt;
	}

#ifdef NKIN_STATES
	if (streamID == 0) {
		NeuroGPUKernel << <currKernelRun, blockDim, TotalSMem, stream0 >> > (stim_d, &d_modelParams[prevRuns*memSizeForModelParams], &d_initStates[prevRuns*memSizeForInitStatae], sim_d, Mat_d, V_d, &VHotsGlobal[prevRuns*memSizeForVHotGlobal], CompDepth, CompFDepth); // RRR	

		CUDA_RT_CALL(hipMemcpyAsync(&VHotsHost[prevRuns*memSizeForVHotGlobal], &VHotsGlobal[prevRuns*memSizeForVHotGlobal], currKernelRun * Nt * sim.NRecSites * stim.NStimuli * sizeof(MYFTYPE), hipMemcpyDeviceToHost, stream0);
		printf("dev id is %d, cudastatus is %s\n", currDevice, cudaStatus);

	}
#endif
#ifndef NKIN_STATES
	//printf("kernel not ran yet\n");
	//NeuroGPUKernel << <currKernelRun, blockDim, TotalSMem >> >(stim_d, d_modelParams, d_modelStates,d_cai, d_ica, d_eca, sim_d, Mat_d, V_d, VHotsGlobal, CompDepth, CompFDepth); // RRR	
	//printf("kernel ran before memcpyasync currkernel run is %d\n", currKernelRun);
	//CUDA_RT_CALL(hipMemcpyAsync(VHotsHost, V_d, currKernelRun * Nt * sim.NRecSites * stim.NStimuli * sizeof(MYFTYPE), hipMemcpyDeviceToHost));


	//printf("done copying*&*&*&*&*&*&*\n");
	//printf ("dev id is %d, cudastatus is %s\n",currDevice,cudaStatus);
#endif


	//currDevice += 1;


	
}

void stEfork2Main(Stim stim, Sim sim, MYFTYPE* ParamsM, MYFTYPE* InitStatesM, HMat& InMat, MYFTYPE* V, MYDTYPE CompDepth, MYDTYPE CompFDepth, int NSets, int* p2pCapableGPUs, int np2p) {
	MYFTYPE *Vhots;
	MYFTYPE **vhots_dev;
	MYDTYPE Nt = stim.Nt;
	printf("in stefork\n");
	vhots_dev = (MYFTYPE**)(malloc(np2p * sizeof(MYFTYPE*)));
	Vhots = (MYFTYPE*)malloc(NSets*Nt*stim.NStimuli*sim.NRecSites * sizeof(MYFTYPE));

	//hipEventCreate(&start);
	//hipEventCreate(&stop);
	HMat Mat_d;
	//hipError_t cudaStatus;
	if (np2p == 0) {
		np2p = 1;
	}
	for (int i = 0; i < np2p; i++) {
		printf("calling initframework p2pCapableGPUs[i] is %d\n", p2pCapableGPUs[i]);
		CUDA_RT_CALL(hipSetDevice(p2pCapableGPUs[i]));
		//hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
		initFrameWork(stim, sim, ParamsM, InitStatesM, InMat, CompDepth, CompFDepth, NSets, Mat_d);
	}
	//printf("in steforkmain p2pCapableGPUs[0] is %d\n", p2pCapableGPUs[0]);
	//CUDA_RT_CALL(hipSetDevice(p2pCapableGPUs[0]));
	//printf("calling initframework dev0\n");


	//RRR sim
	MYDTYPE prevRuns = 0;
	MYDTYPE currRun;
	if (NSets > np2p) {
		currRun = ceil(NSets / np2p);
	}
	else { currRun = NSets; };

	printf("done initframework dev0 curr Kernel is %d\n", currRun);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);

	for (int i = 0; i < np2p; i++) {
		if (prevRuns >= NSets)  break;
		CUDA_RT_CALL(hipSetDevice(p2pCapableGPUs[i]));
		printf("calling kernel dev%d\n", p2pCapableGPUs[i]);
		
		vhots_dev[i] = (MYFTYPE*)malloc(currRun*Nt*stim.NStimuli*sim.NRecSites * sizeof(MYFTYPE));
		callKernel(stim, sim, ParamsM, InitStatesM, Mat_d, V, CompDepth, CompFDepth, prevRuns, currRun, vhots_dev[i]);
		prevRuns += currRun;

	}
	for (int i = 0; i < np2p; i++) {
		CUDA_RT_CALL(hipSetDevice(p2pCapableGPUs[i]));
		CUDA_RT_CALL(hipDeviceSynchronize());
		printf("done synch%d\n", p2pCapableGPUs[i]);
		if (NSets <np2p) {
			printf("nsets >p2pdevs");
			vhots_dev[0] = transposeMat(vhots_dev[0], currRun, Nt);
			Vhots = vhots_dev[0];
		}
		else {
			vhots_dev[i] = transposeMat(vhots_dev[i], currRun, Nt);
			memcpy(&Vhots[(currRun*i)*stim.NStimuli*Nt*sim.NRecSites], vhots_dev[i], stim.NStimuli*Nt*sim.NRecSites*currRun * sizeof(MYFTYPE));
		}
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("it took %f ms\n", milliseconds);
	FILE *file = fopen(TIMES_FN, "w");
	if (file) {
		fprintf(file, "%d,%f\n", NSets, milliseconds);
	}
	else {
		printf("ERR SaveArrayToFile %s\n", TIMES_FN);
	}
	fclose(file);

	SaveArrayToFile(VHOT_OUT_FN_P, NSets*Nt*stim.NStimuli*sim.NRecSites, Vhots);



}



